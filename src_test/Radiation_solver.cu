#include "hip/hip_runtime.h"
/*
 * This file is imported from MicroHH (https://github.com/earth-system-radiation/earth-system-radiation)
 * and is adapted for the testing of the C++ interface to the
 * RTE+RRTMGP radiation code.
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/algorithm/string.hpp>
#include <cmath>
#include <numeric>

#include "Radiation_solver.h"
#include "Status.h"
#include "Netcdf_interface.h"

#include "Array.h"
#include "Gas_concs.h"
#include "Gas_optics_rrtmgp.h"
#include "Optical_props.h"
#include "Source_functions.h"
#include "Fluxes.h"
#include "Rte_lw.h"
#include "Rte_sw.h"
#include "subset_kernel_launcher_cuda.h"

#include <hip/hip_runtime_api.h>
namespace
{
    template<typename TF>__global__
    void scaling_to_subset_kernel(
            const int ncol, const int ngpt, TF* __restrict__ toa_src, const TF* __restrict__ tsi_scaling)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int igpt = blockIdx.y*blockDim.y + threadIdx.y;
        if ( ( icol < ncol) && (igpt < ngpt) )
        {
            const int idx = icol + igpt*ncol;
            toa_src[idx] *= tsi_scaling[icol];
        }
    }

    template<typename TF>
    void scaling_to_subset(
            const int ncol, const int ngpt, Array_gpu<TF,2>& toa_src, const Array_gpu<TF,1>& tsi_scaling)
    {
        const int block_col = 16;
        const int block_gpt = 16;
        const int grid_col  = ncol/block_col + (ncol%block_col > 0);
        const int grid_gpt  = ngpt/block_gpt + (ngpt%block_gpt > 0);
        dim3 grid_gpu(grid_col, grid_gpt);
        dim3 block_gpu(block_col, block_gpt);
        scaling_to_subset_kernel<<<grid_gpu, block_gpu>>>(
            ncol, ngpt, toa_src.ptr(), tsi_scaling.ptr());
    }


    std::vector<std::string> get_variable_string(
            const std::string& var_name,
            std::vector<int> i_count,
            Netcdf_handle& input_nc,
            const int string_len,
            bool trim=true)
    {
        // Multiply all elements in i_count.
        int total_count = std::accumulate(i_count.begin(), i_count.end(), 1, std::multiplies<>());

        // Add the string length as the rightmost dimension.
        i_count.push_back(string_len);

        // Read the entire char array;
        std::vector<char> var_char;
        var_char = input_nc.get_variable<char>(var_name, i_count);

        std::vector<std::string> var;

        for (int n=0; n<total_count; ++n)
        {
            std::string s(var_char.begin()+n*string_len, var_char.begin()+(n+1)*string_len);
            if (trim)
                boost::trim(s);
            var.push_back(s);
        }

        return var;
    }

    template<typename TF>
    Gas_optics_rrtmgp_gpu<TF> load_and_init_gas_optics(
            const Gas_concs_gpu<TF>& gas_concs,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(coef_file, Netcdf_mode::Read);

        // Read k-distribution information.
        int n_temps = coef_nc.get_dimension_size("temperature");
        int n_press = coef_nc.get_dimension_size("pressure");
        int n_absorbers = coef_nc.get_dimension_size("absorber");

        // CvH: I hardcode the value to 32 now, because coef files
        // CvH: changed dimension name inconsistently.
        // int n_char = coef_nc.get_dimension_size("string_len");
        constexpr int n_char = 32;

        int n_minorabsorbers = coef_nc.get_dimension_size("minor_absorber");
        int n_extabsorbers = coef_nc.get_dimension_size("absorber_ext");
        int n_mixingfracs = coef_nc.get_dimension_size("mixing_fraction");
        int n_layers = coef_nc.get_dimension_size("atmos_layer");
        int n_bnds = coef_nc.get_dimension_size("bnd");
        int n_gpts = coef_nc.get_dimension_size("gpt");
        int n_pairs = coef_nc.get_dimension_size("pair");
        int n_minor_absorber_intervals_lower = coef_nc.get_dimension_size("minor_absorber_intervals_lower");
        int n_minor_absorber_intervals_upper = coef_nc.get_dimension_size("minor_absorber_intervals_upper");
        int n_contributors_lower = coef_nc.get_dimension_size("contributors_lower");
        int n_contributors_upper = coef_nc.get_dimension_size("contributors_upper");

        // Read gas names.
        Array<std::string,1> gas_names(
                get_variable_string("gas_names", {n_absorbers}, coef_nc, n_char, true), {n_absorbers});

        Array<int,3> key_species(
                coef_nc.get_variable<int>("key_species", {n_bnds, n_layers, 2}),
                {2, n_layers, n_bnds});
        Array<TF,2> band_lims(coef_nc.get_variable<TF>("bnd_limits_wavenumber", {n_bnds, 2}), {2, n_bnds});
        Array<int,2> band2gpt(coef_nc.get_variable<int>("bnd_limits_gpt", {n_bnds, 2}), {2, n_bnds});
        Array<TF,1> press_ref(coef_nc.get_variable<TF>("press_ref", {n_press}), {n_press});
        Array<TF,1> temp_ref(coef_nc.get_variable<TF>("temp_ref", {n_temps}), {n_temps});

        TF temp_ref_p = coef_nc.get_variable<TF>("absorption_coefficient_ref_P");
        TF temp_ref_t = coef_nc.get_variable<TF>("absorption_coefficient_ref_T");
        TF press_ref_trop = coef_nc.get_variable<TF>("press_ref_trop");

        Array<TF,3> kminor_lower(
                coef_nc.get_variable<TF>("kminor_lower", {n_temps, n_mixingfracs, n_contributors_lower}),
                {n_contributors_lower, n_mixingfracs, n_temps});
        Array<TF,3> kminor_upper(
                coef_nc.get_variable<TF>("kminor_upper", {n_temps, n_mixingfracs, n_contributors_upper}),
                {n_contributors_upper, n_mixingfracs, n_temps});

        Array<std::string,1> gas_minor(get_variable_string("gas_minor", {n_minorabsorbers}, coef_nc, n_char),
                                       {n_minorabsorbers});

        Array<std::string,1> identifier_minor(
                get_variable_string("identifier_minor", {n_minorabsorbers}, coef_nc, n_char), {n_minorabsorbers});

        Array<std::string,1> minor_gases_lower(
                get_variable_string("minor_gases_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> minor_gases_upper(
                get_variable_string("minor_gases_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,2> minor_limits_gpt_lower(
                coef_nc.get_variable<int>("minor_limits_gpt_lower", {n_minor_absorber_intervals_lower, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_lower});
        Array<int,2> minor_limits_gpt_upper(
                coef_nc.get_variable<int>("minor_limits_gpt_upper", {n_minor_absorber_intervals_upper, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_upper});

        Array<BOOL_TYPE,1> minor_scales_with_density_lower(
                coef_nc.get_variable<BOOL_TYPE>("minor_scales_with_density_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<BOOL_TYPE,1> minor_scales_with_density_upper(
                coef_nc.get_variable<BOOL_TYPE>("minor_scales_with_density_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<BOOL_TYPE,1> scale_by_complement_lower(
                coef_nc.get_variable<BOOL_TYPE>("scale_by_complement_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<BOOL_TYPE,1> scale_by_complement_upper(
                coef_nc.get_variable<BOOL_TYPE>("scale_by_complement_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<std::string,1> scaling_gas_lower(
                get_variable_string("scaling_gas_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> scaling_gas_upper(
                get_variable_string("scaling_gas_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,1> kminor_start_lower(
                coef_nc.get_variable<int>("kminor_start_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<int,1> kminor_start_upper(
                coef_nc.get_variable<int>("kminor_start_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<TF,3> vmr_ref(
                coef_nc.get_variable<TF>("vmr_ref", {n_temps, n_extabsorbers, n_layers}),
                {n_layers, n_extabsorbers, n_temps});

        Array<TF,4> kmajor(
                coef_nc.get_variable<TF>("kmajor", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                {n_gpts, n_mixingfracs, n_press+1, n_temps});

        // Keep the size at zero, if it does not exist.
        Array<TF,3> rayl_lower;
        Array<TF,3> rayl_upper;

        if (coef_nc.variable_exists("rayl_lower"))
        {
            rayl_lower.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_upper.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_lower = coef_nc.get_variable<TF>("rayl_lower", {n_temps, n_mixingfracs, n_gpts});
            rayl_upper = coef_nc.get_variable<TF>("rayl_upper", {n_temps, n_mixingfracs, n_gpts});
        }

        // Is it really LW if so read these variables as well.
        if (coef_nc.variable_exists("totplnk"))
        {
            int n_internal_sourcetemps = coef_nc.get_dimension_size("temperature_Planck");

            Array<TF,2> totplnk(
                    coef_nc.get_variable<TF>( "totplnk", {n_bnds, n_internal_sourcetemps}),
                    {n_internal_sourcetemps, n_bnds});
            Array<TF,4> planck_frac(
                    coef_nc.get_variable<TF>("plank_fraction", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                    {n_gpts, n_mixingfracs, n_press+1, n_temps});

            // Construct the k-distribution.
            return Gas_optics_rrtmgp_gpu<TF>(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    totplnk,
                    planck_frac,
                    rayl_lower,
                    rayl_upper);
        }
        else
        {
            Array<TF,1> solar_src_quiet(
                    coef_nc.get_variable<TF>("solar_source_quiet", {n_gpts}), {n_gpts});
            Array<TF,1> solar_src_facular(
                    coef_nc.get_variable<TF>("solar_source_facular", {n_gpts}), {n_gpts});
            Array<TF,1> solar_src_sunspot(
                    coef_nc.get_variable<TF>("solar_source_sunspot", {n_gpts}), {n_gpts});

            TF tsi = coef_nc.get_variable<TF>("tsi_default");
            TF mg_index = coef_nc.get_variable<TF>("mg_default");
            TF sb_index = coef_nc.get_variable<TF>("sb_default");

            return Gas_optics_rrtmgp_gpu<TF>(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    solar_src_quiet,
                    solar_src_facular,
                    solar_src_sunspot,
                    tsi,
                    mg_index,
                    sb_index,
                    rayl_lower,
                    rayl_upper);
        }
        // End reading of k-distribution.
    }

    template<typename TF>
    Cloud_optics_gpu<TF> load_and_init_cloud_optics(
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(coef_file, Netcdf_mode::Read);

        // Read look-up table coefficient dimensions
        int n_band     = coef_nc.get_dimension_size("nband");
        int n_rghice   = coef_nc.get_dimension_size("nrghice");
        int n_size_liq = coef_nc.get_dimension_size("nsize_liq");
        int n_size_ice = coef_nc.get_dimension_size("nsize_ice");

        Array<TF,2> band_lims_wvn(coef_nc.get_variable<TF>("bnd_limits_wavenumber", {n_band, 2}), {2, n_band});

        // Read look-up table constants.
        TF radliq_lwr = coef_nc.get_variable<TF>("radliq_lwr");
        TF radliq_upr = coef_nc.get_variable<TF>("radliq_upr");
        TF radliq_fac = coef_nc.get_variable<TF>("radliq_fac");

        TF radice_lwr = coef_nc.get_variable<TF>("radice_lwr");
        TF radice_upr = coef_nc.get_variable<TF>("radice_upr");
        TF radice_fac = coef_nc.get_variable<TF>("radice_fac");

        Array<TF,2> lut_extliq(
                coef_nc.get_variable<TF>("lut_extliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<TF,2> lut_ssaliq(
                coef_nc.get_variable<TF>("lut_ssaliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<TF,2> lut_asyliq(
                coef_nc.get_variable<TF>("lut_asyliq", {n_band, n_size_liq}), {n_size_liq, n_band});

        Array<TF,3> lut_extice(
                coef_nc.get_variable<TF>("lut_extice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<TF,3> lut_ssaice(
                coef_nc.get_variable<TF>("lut_ssaice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<TF,3> lut_asyice(
                coef_nc.get_variable<TF>("lut_asyice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});

        return Cloud_optics_gpu<TF>(
                band_lims_wvn,
                radliq_lwr, radliq_upr, radliq_fac,
                radice_lwr, radice_upr, radice_fac,
                lut_extliq, lut_ssaliq, lut_asyliq,
                lut_extice, lut_ssaice, lut_asyice);
    }
}

template<typename TF>
Radiation_solver_longwave<TF>::Radiation_solver_longwave(
        const Gas_concs_gpu<TF>& gas_concs,
        const std::string& file_name_gas,
        const std::string& file_name_cloud)
{
    // Construct the gas optics classes for the solver.
    this->kdist_gpu = std::make_unique<Gas_optics_rrtmgp_gpu<TF>>(
            load_and_init_gas_optics<TF>(gas_concs, file_name_gas));

    this->cloud_optics_gpu = std::make_unique<Cloud_optics_gpu<TF>>(
            load_and_init_cloud_optics<TF>(file_name_cloud));
}

template<typename TF>
void Radiation_solver_longwave<TF>::solve_gpu(
        const bool switch_fluxes,
        const bool switch_cloud_optics,
        const bool switch_output_optical,
        const bool switch_output_bnd_fluxes,
        const Gas_concs_gpu<TF>& gas_concs,
        const Array_gpu<TF,2>& p_lay, const Array_gpu<TF,2>& p_lev,
        const Array_gpu<TF,2>& t_lay, const Array_gpu<TF,2>& t_lev,
        const Array_gpu<TF,2>& col_dry,
        const Array_gpu<TF,1>& t_sfc, const Array_gpu<TF,2>& emis_sfc,
        const Array_gpu<TF,2>& lwp, const Array_gpu<TF,2>& iwp,
        const Array_gpu<TF,2>& rel, const Array_gpu<TF,2>& rei,
        Array_gpu<TF,3>& tau, Array_gpu<TF,3>& lay_source,
        Array_gpu<TF,3>& lev_source_inc, Array_gpu<TF,3>& lev_source_dec, Array_gpu<TF,2>& sfc_source,
        Array_gpu<TF,2>& lw_flux_up, Array_gpu<TF,2>& lw_flux_dn, Array_gpu<TF,2>& lw_flux_net,
        Array_gpu<TF,3>& lw_bnd_flux_up, Array_gpu<TF,3>& lw_bnd_flux_dn, Array_gpu<TF,3>& lw_bnd_flux_net) const
{
    const int n_col = p_lay.dim(1);
    const int n_lay = p_lay.dim(2);
    const int n_lev = p_lev.dim(2);
    const int n_gpt = this->kdist_gpu->get_ngpt();
    const int n_bnd = this->kdist_gpu->get_nband();

    const BOOL_TYPE top_at_1 = p_lay({1, 1}) < p_lay({1, n_lay});

    constexpr int n_col_block = 16;

    // Read the sources and create containers for the substeps.
    int n_blocks = n_col / n_col_block;
    int n_col_block_residual = n_col % n_col_block;

    std::unique_ptr<Optical_props_arry_gpu<TF>> optical_props_subset;
    std::unique_ptr<Optical_props_arry_gpu<TF>> optical_props_residual;

    optical_props_subset = std::make_unique<Optical_props_1scl_gpu<TF>>(n_col_block, n_lay, *kdist_gpu);

    std::unique_ptr<Source_func_lw_gpu<TF>> sources_subset;
    std::unique_ptr<Source_func_lw_gpu<TF>> sources_residual;

    sources_subset = std::make_unique<Source_func_lw_gpu<TF>>(n_col_block, n_lay, *kdist_gpu);

    if (n_col_block_residual > 0)
    {
        optical_props_residual = std::make_unique<Optical_props_1scl_gpu<TF>>(n_col_block_residual, n_lay, *kdist_gpu);
        sources_residual = std::make_unique<Source_func_lw_gpu<TF>>(n_col_block_residual, n_lay, *kdist_gpu);
    }

    std::unique_ptr<Optical_props_1scl_gpu<TF>> cloud_optical_props_subset;
    std::unique_ptr<Optical_props_1scl_gpu<TF>> cloud_optical_props_residual;

    if (switch_cloud_optics)
    {
        cloud_optical_props_subset = std::make_unique<Optical_props_1scl_gpu<TF>>(n_col_block, n_lay, *cloud_optics_gpu);
        if (n_col_block_residual > 0)
            cloud_optical_props_residual = std::make_unique<Optical_props_1scl_gpu<TF>>(n_col_block_residual, n_lay, *cloud_optics_gpu);
    }

    // Lambda function for solving optical properties subset.
    auto call_kernels = [&](
            const int col_s_in, const int col_e_in,
            std::unique_ptr<Optical_props_arry_gpu<TF>>& optical_props_subset_in,
            std::unique_ptr<Optical_props_1scl_gpu<TF>>& cloud_optical_props_subset_in,
            Source_func_lw_gpu<TF>& sources_subset_in,
            const Array_gpu<TF,2>& emis_sfc_subset_in,
            Fluxes_broadband_gpu<TF>& fluxes,
            Fluxes_broadband_gpu<TF>& bnd_fluxes)
    {
        const int n_col_in = col_e_in - col_s_in + 1;
        Gas_concs_gpu<TF> gas_concs_subset(gas_concs, col_s_in, n_col_in);

        auto p_lev_subset = p_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }});

        Array_gpu<TF,2> col_dry_subset({n_col_in, n_lay});
        if (col_dry.size() == 0)
            Gas_optics_rrtmgp_gpu<TF>::get_col_dry(col_dry_subset, gas_concs_subset.get_vmr("h2o"), p_lev_subset);
        else
            col_dry_subset = col_dry.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});

        kdist_gpu->gas_optics(
                p_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                p_lev_subset,
                t_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                t_sfc.subset({{ {col_s_in, col_e_in} }}),
                gas_concs_subset,
                optical_props_subset_in,
                sources_subset_in,
                col_dry_subset,
                t_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }}) );

        if (switch_cloud_optics)
        {
            cloud_optics_gpu->cloud_optics(
                    lwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    iwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    rel.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    rei.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    *cloud_optical_props_subset_in);

            // cloud->delta_scale();

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_1scl_gpu<double>&>(*optical_props_subset_in),
                    dynamic_cast<Optical_props_1scl_gpu<double>&>(*cloud_optical_props_subset_in));
        }

        // Store the optical properties, if desired.
        if (switch_output_optical)
        {
            subset_kernel_launcher_cuda::get_from_subset(
                    n_col, n_lay, n_gpt, n_col_in, col_s_in, tau, lay_source, lev_source_inc, lev_source_dec,
                    optical_props_subset_in->get_tau(), sources_subset_in.get_lay_source(),
                    sources_subset_in.get_lev_source_inc(), sources_subset_in.get_lev_source_dec());

            subset_kernel_launcher_cuda::get_from_subset(
                    n_col, n_gpt, n_col_in, col_s_in, sfc_source, sources_subset_in.get_sfc_source());
        }

        if (!switch_fluxes)
            return;

        Array_gpu<TF,3> gpt_flux_up({n_col_in, n_lev, n_gpt});
        Array_gpu<TF,3> gpt_flux_dn({n_col_in, n_lev, n_gpt});

        constexpr int n_ang = 1;

        Rte_lw_gpu<TF>::rte_lw(
                optical_props_subset_in,
                top_at_1,
                sources_subset_in,
                emis_sfc_subset_in,
                Array_gpu<TF,2>(), // Add an empty array, no inc_flux.
                gpt_flux_up, gpt_flux_dn,
                n_ang);
        fluxes.reduce(gpt_flux_up, gpt_flux_dn, optical_props_subset_in, top_at_1);

        // Copy the data to the output.
        subset_kernel_launcher_cuda::get_from_subset(
                n_col, n_lev, n_col_in, col_s_in, lw_flux_up, lw_flux_dn, lw_flux_net,
                fluxes.get_flux_up(), fluxes.get_flux_dn(), fluxes.get_flux_net());


        if (switch_output_bnd_fluxes)
        {
            bnd_fluxes.reduce(gpt_flux_up, gpt_flux_dn, optical_props_subset_in, top_at_1);

            subset_kernel_launcher_cuda::get_from_subset(
                    n_col, n_lev, n_bnd, n_col_in, col_s_in, lw_bnd_flux_up, lw_bnd_flux_dn, lw_bnd_flux_net,
                    bnd_fluxes.get_bnd_flux_up(), bnd_fluxes.get_bnd_flux_dn(), bnd_fluxes.get_bnd_flux_net());

        }
    };

    for (int b=1; b<=n_blocks; ++b)
    {
        const int col_s = (b-1) * n_col_block + 1;
        const int col_e =  b    * n_col_block;

        Array_gpu<TF,2> emis_sfc_subset = emis_sfc.subset({{ {1, n_bnd}, {col_s, col_e} }});

        std::unique_ptr<Fluxes_broadband_gpu<TF>> fluxes_subset =
                std::make_unique<Fluxes_broadband_gpu<TF>>(n_col_block, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu<TF>> bnd_fluxes_subset =
                std::make_unique<Fluxes_byband_gpu<TF>>(n_col_block, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_subset,
                cloud_optical_props_subset,
                *sources_subset,
                emis_sfc_subset,
                *fluxes_subset,
                *bnd_fluxes_subset);
    }

    if (n_col_block_residual > 0)
    {
        const int col_s = n_col - n_col_block_residual + 1;
        const int col_e = n_col;

        Array_gpu<TF,2> emis_sfc_residual = emis_sfc.subset({{ {1, n_bnd}, {col_s, col_e} }});
        std::unique_ptr<Fluxes_broadband_gpu<TF>> fluxes_residual =
                std::make_unique<Fluxes_broadband_gpu<TF>>(n_col_block_residual, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu<TF>> bnd_fluxes_residual =
                std::make_unique<Fluxes_byband_gpu<TF>>(n_col_block_residual, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_residual,
                cloud_optical_props_residual,
                *sources_residual,
                emis_sfc_residual,
                *fluxes_residual,
                *bnd_fluxes_residual);
    }
}

template<typename TF>
Radiation_solver_shortwave<TF>::Radiation_solver_shortwave(
        const Gas_concs_gpu<TF>& gas_concs,
        const std::string& file_name_gas,
        const std::string& file_name_cloud)
{
    // Construct the gas optics classes for the solver.
    this->kdist_gpu = std::make_unique<Gas_optics_rrtmgp_gpu<TF>>(
            load_and_init_gas_optics<TF>(gas_concs, file_name_gas));

    this->cloud_optics_gpu = std::make_unique<Cloud_optics_gpu<TF>>(
            load_and_init_cloud_optics<TF>(file_name_cloud));
}


template<typename TF>
void Radiation_solver_shortwave<TF>::solve_gpu(
        const bool switch_fluxes,
        const bool switch_cloud_optics,
        const bool switch_output_optical,
        const bool switch_output_bnd_fluxes,
        const Gas_concs_gpu<TF>& gas_concs,
        const Array_gpu<TF,2>& p_lay, const Array_gpu<TF,2>& p_lev,
        const Array_gpu<TF,2>& t_lay, const Array_gpu<TF,2>& t_lev,
        const Array_gpu<TF,2>& col_dry,
        const Array_gpu<TF,2>& sfc_alb_dir, const Array_gpu<TF,2>& sfc_alb_dif,
        const Array_gpu<TF,1>& tsi_scaling, const Array_gpu<TF,1>& mu0,
        const Array_gpu<TF,2>& lwp, const Array_gpu<TF,2>& iwp,
        const Array_gpu<TF,2>& rel, const Array_gpu<TF,2>& rei,
        Array_gpu<TF,3>& tau, Array_gpu<TF,3>& ssa, Array_gpu<TF,3>& g,
        Array_gpu<TF,2>& toa_src,
        Array_gpu<TF,2>& sw_flux_up, Array_gpu<TF,2>& sw_flux_dn,
        Array_gpu<TF,2>& sw_flux_dn_dir, Array_gpu<TF,2>& sw_flux_net,
        Array_gpu<TF,3>& sw_bnd_flux_up, Array_gpu<TF,3>& sw_bnd_flux_dn,
        Array_gpu<TF,3>& sw_bnd_flux_dn_dir, Array_gpu<TF,3>& sw_bnd_flux_net) const
{
    const int n_col = p_lay.dim(1);
    const int n_lay = p_lay.dim(2);
    const int n_lev = p_lev.dim(2);
    const int n_gpt = this->kdist_gpu->get_ngpt();
    const int n_bnd = this->kdist_gpu->get_nband();

    const BOOL_TYPE top_at_1 = p_lay({1, 1}) < p_lay({1, n_lay});

    constexpr int n_col_block = 16;

    // Read the sources and create containers for the substeps.
    int n_blocks = n_col / n_col_block;
    int n_col_block_residual = n_col % n_col_block;

    std::unique_ptr<Optical_props_arry_gpu<TF>> optical_props_subset;
    std::unique_ptr<Optical_props_arry_gpu<TF>> optical_props_residual;

    optical_props_subset = std::make_unique<Optical_props_2str_gpu<TF>>(n_col_block, n_lay, *kdist_gpu);
    if (n_col_block_residual > 0)
        optical_props_residual = std::make_unique<Optical_props_2str_gpu<TF>>(n_col_block_residual, n_lay, *kdist_gpu);

    std::unique_ptr<Optical_props_2str_gpu<TF>> cloud_optical_props_subset;
    std::unique_ptr<Optical_props_2str_gpu<TF>> cloud_optical_props_residual;

    if (switch_cloud_optics)
    {
        cloud_optical_props_subset = std::make_unique<Optical_props_2str_gpu<TF>>(n_col_block, n_lay, *cloud_optics_gpu);
        if (n_col_block_residual > 0)
            cloud_optical_props_residual = std::make_unique<Optical_props_2str_gpu<TF>>(n_col_block_residual, n_lay, *cloud_optics_gpu);
    }

    // Lambda function for solving optical properties subset.
    auto call_kernels = [&, n_col, n_lay, n_lev, n_gpt, n_bnd](
            const int col_s_in, const int col_e_in,
            std::unique_ptr<Optical_props_arry_gpu<TF>>& optical_props_subset_in,
            std::unique_ptr<Optical_props_2str_gpu<TF>>& cloud_optical_props_subset_in,
            Fluxes_broadband_gpu<TF>& fluxes,
            Fluxes_broadband_gpu<TF>& bnd_fluxes)
    {
        const int n_col_in = col_e_in - col_s_in + 1;
        Gas_concs_gpu<TF> gas_concs_subset(gas_concs, col_s_in, n_col_in);

        auto p_lev_subset = p_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }});

        Array_gpu<TF,2> col_dry_subset({n_col_in, n_lay});
        if (col_dry.size() == 0)
            Gas_optics_rrtmgp_gpu<TF>::get_col_dry(col_dry_subset, gas_concs_subset.get_vmr("h2o"), p_lev_subset);
        else
            col_dry_subset = col_dry.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});

        Array_gpu<TF,2> toa_src_subset({n_col_in, n_gpt});
        kdist_gpu->gas_optics(
                  p_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                  p_lev_subset,
                  t_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                  gas_concs_subset,
                  optical_props_subset_in,
                  toa_src_subset,
                  col_dry_subset);
        auto tsi_scaling_subset = tsi_scaling.subset({{ {col_s_in, col_e_in} }});
        scaling_to_subset(n_col_in, n_gpt, toa_src_subset, tsi_scaling_subset);
        if (switch_cloud_optics)
        {
            Array<int,2> cld_mask_liq({n_col_in, n_lay});
            Array<int,2> cld_mask_ice({n_col_in, n_lay});

            cloud_optics_gpu->cloud_optics(
                    lwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    iwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    rel.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    rei.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    *cloud_optical_props_subset_in);

            cloud_optical_props_subset_in->delta_scale();

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_2str_gpu<double>&>(*optical_props_subset_in),
                    dynamic_cast<Optical_props_2str_gpu<double>&>(*cloud_optical_props_subset_in));

        }

        // Store the optical properties, if desired.
        if (switch_output_optical)
        {
            subset_kernel_launcher_cuda::get_from_subset(
                    n_col, n_lay, n_gpt, n_col_in, col_s_in, tau, ssa, g, optical_props_subset_in->get_tau(),
                     optical_props_subset_in->get_ssa(),  optical_props_subset_in->get_g());

            subset_kernel_launcher_cuda::get_from_subset(
                    n_col, n_gpt, n_col_in, col_s_in, toa_src, toa_src_subset);
        }
        if (!switch_fluxes)
            return;

        Array_gpu<TF,3> gpt_flux_up    ({n_col_in, n_lev, n_gpt});
        Array_gpu<TF,3> gpt_flux_dn    ({n_col_in, n_lev, n_gpt});
        Array_gpu<TF,3> gpt_flux_dn_dir({n_col_in, n_lev, n_gpt});
        Rte_sw_gpu<TF>::rte_sw(
                optical_props_subset_in,
                top_at_1,
                mu0.subset({{ {col_s_in, col_e_in} }}),
                toa_src_subset,
                sfc_alb_dir.subset({{ {1, n_bnd}, {col_s_in, col_e_in} }}),
                sfc_alb_dif.subset({{ {1, n_bnd}, {col_s_in, col_e_in} }}),
                Array_gpu<TF,2>(), // Add an empty array, no inc_flux.
                gpt_flux_up,
                gpt_flux_dn,
                gpt_flux_dn_dir);

        fluxes.reduce(gpt_flux_up, gpt_flux_dn, gpt_flux_dn_dir, optical_props_subset_in, top_at_1);

        // Copy the data to the output.
        subset_kernel_launcher_cuda::get_from_subset(
                n_col, n_lev, n_col_in, col_s_in, sw_flux_up, sw_flux_dn, sw_flux_dn_dir, sw_flux_net,
                fluxes.get_flux_up(), fluxes.get_flux_dn(), fluxes.get_flux_dn_dir(), fluxes.get_flux_net());

        if (switch_output_bnd_fluxes)
        {
            bnd_fluxes.reduce(gpt_flux_up, gpt_flux_dn, optical_props_subset_in, top_at_1);

            subset_kernel_launcher_cuda::get_from_subset(
                    n_col, n_lev, n_bnd, n_col_in, col_s_in, sw_bnd_flux_up, sw_bnd_flux_dn, sw_bnd_flux_dn_dir, sw_bnd_flux_net,
                    bnd_fluxes.get_bnd_flux_up(), bnd_fluxes.get_bnd_flux_dn(), bnd_fluxes.get_bnd_flux_dn_dir(), bnd_fluxes.get_bnd_flux_net());
        }
    };


    for (int b=1; b<=n_blocks; ++b)
    {
        const int col_s = (b-1) * n_col_block + 1;
        const int col_e =  b    * n_col_block;

        std::unique_ptr<Fluxes_broadband_gpu<TF>> fluxes_subset =
                std::make_unique<Fluxes_broadband_gpu<TF>>(n_col_block, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu<TF>> bnd_fluxes_subset =
                std::make_unique<Fluxes_byband_gpu<TF>>(n_col_block, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_subset,
                cloud_optical_props_subset,
                *fluxes_subset,
                *bnd_fluxes_subset);
    }

    if (n_col_block_residual > 0)
    {
        const int col_s = n_col - n_col_block_residual + 1;
        const int col_e = n_col;

        std::unique_ptr<Fluxes_broadband_gpu<TF>> fluxes_residual =
                std::make_unique<Fluxes_broadband_gpu<TF>>(n_col_block_residual, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu<TF>> bnd_fluxes_residual =
                std::make_unique<Fluxes_byband_gpu<TF>>(n_col_block_residual, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_residual,
                cloud_optical_props_residual,
                *fluxes_residual,
                *bnd_fluxes_residual);
    }
}

#ifdef FLOAT_SINGLE_RRTMGP
template class Radiation_solver_longwave<float>;
template class Radiation_solver_shortwave<float>;
#else
template class Radiation_solver_longwave<double>;
template class Radiation_solver_shortwave<double>;
#endif
