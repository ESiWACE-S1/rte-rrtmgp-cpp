#include "hip/hip_runtime.h"
/*
 * This file is part of a C++ interface to the Radiative Transfer for Energetics (RTE)
 * and Rapid Radiative Transfer Model for GCM applications Parallel (RRTMGP).
 *
 * The original code is found at https://github.com/earth-system-radiation/rte-rrtmgp.
 *
 * Contacts: Robert Pincus and Eli Mlawer
 * email: rrtmgp@aer.com
 *
 * Copyright 2015-2020,  Atmospheric and Environmental Research and
 * Regents of the University of Colorado.  All right reserved.
 *
 * This C++ interface can be downloaded from https://github.com/earth-system-radiation/rte-rrtmgp-cpp
 *
 * Contact: Chiel van Heerwaarden
 * email: chiel.vanheerwaarden@wur.nl
 *
 * Copyright 2020, Wageningen University & Research.
 *
 * Use and duplication is permitted under the terms of the
 * BSD 3-clause license, see http://opensource.org/licenses/BSD-3-Clause
 *
 */

#include "Rte_sw.h"
#include "Array.h"
#include "Optical_props.h"
#include "Fluxes.h"
#include "rrtmgp_kernels.h"
// CUDA TEST
#include "rte_kernel_launcher_cuda.h"
// END CUDA TEST

namespace
{
    template<typename TF>__global__
    void expand_and_transpose_kernel(
        const int ncol, const int nbnd, const int* __restrict__ limits,
        TF* __restrict__ arr_out, const TF* __restrict__ arr_in)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ibnd = blockIdx.y*blockDim.y + threadIdx.y;

        if ( ( icol < ncol) && (ibnd < nbnd) )
        {
            const int gpt_start = limits[2*ibnd] - 1;
            const int gpt_end = limits[2*ibnd+1];

            for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
            {
                const int idx_in = ibnd + icol*nbnd;
                const int idx_out = icol + igpt*ncol;

                arr_out[idx_out] = arr_in[idx_in];
            }
        }
    }
}

//namespace rrtmgp_kernel_launcher
//{
//    template<typename TF>
//    void apply_BC(
//            int ncol, int nlay, int ngpt,
//            BOOL_TYPE top_at_1, Array<TF,3>& gpt_flux_dn)
//    {
//        rrtmgp_kernels::apply_BC_0(
//                &ncol, &nlay, &ngpt,
//                &top_at_1, gpt_flux_dn.ptr());
//    }
//
//    template<typename TF>
//    void apply_BC(
//            int ncol, int nlay, int ngpt, BOOL_TYPE top_at_1,
//            const Array<TF,2>& inc_flux, Array<TF,3>& gpt_flux_dn)
//    {
//        rrtmgp_kernels::apply_BC_gpt(
//                &ncol, &nlay, &ngpt, &top_at_1,
//                const_cast<TF*>(inc_flux.ptr()), gpt_flux_dn.ptr());
//    }
//
//    template<typename TF>
//    void apply_BC(
//            int ncol, int nlay, int ngpt, BOOL_TYPE top_at_1,
//            const Array<TF,2>& inc_flux,
//            const Array<TF,1>& factor,
//            Array<TF,3>& gpt_flux)
//    {
//        rrtmgp_kernels::apply_BC_factor(
//                &ncol, &nlay, &ngpt,
//                &top_at_1,
//                const_cast<TF*>(inc_flux.ptr()),
//                const_cast<TF*>(factor.ptr()),
//                gpt_flux.ptr());
//    }
//
//    template<typename TF>
//    void sw_solver_2stream(
//            int ncol, int nlay, int ngpt, BOOL_TYPE top_at_1,
//            const Array<TF,3>& tau,
//            const Array<TF,3>& ssa,
//            const Array<TF,3>& g,
//            const Array<TF,1>& mu0,
//            const Array<TF,2>& sfc_alb_dir_gpt, const Array<TF,2>& sfc_alb_dif_gpt,
//            Array<TF,3>& gpt_flux_up, Array<TF,3>& gpt_flux_dn, Array<TF,3>& gpt_flux_dir)
//    {
//        rrtmgp_kernels::sw_solver_2stream(
//                &ncol, &nlay, &ngpt, &top_at_1,
//                const_cast<TF*>(tau.ptr()),
//                const_cast<TF*>(ssa.ptr()),
//                const_cast<TF*>(g  .ptr()),
//                const_cast<TF*>(mu0.ptr()),
//                const_cast<TF*>(sfc_alb_dir_gpt.ptr()),
//                const_cast<TF*>(sfc_alb_dif_gpt.ptr()),
//                gpt_flux_up.ptr(), gpt_flux_dn.ptr(), gpt_flux_dir.ptr());
//    }

template<typename TF>
void Rte_sw_gpu<TF>::rte_sw(
        const std::unique_ptr<Optical_props_arry_gpu<TF>>& optical_props,
        const BOOL_TYPE top_at_1,
        const Array_gpu<TF,1>& mu0,
        const Array_gpu<TF,2>& inc_flux_dir,
        const Array_gpu<TF,2>& sfc_alb_dir,
        const Array_gpu<TF,2>& sfc_alb_dif,
        const Array_gpu<TF,2>& inc_flux_dif,
        Array_gpu<TF,3>& gpt_flux_up,
        Array_gpu<TF,3>& gpt_flux_dn,
        Array_gpu<TF,3>& gpt_flux_dir)
{
    const int ncol = optical_props->get_ncol();
    const int nlay = optical_props->get_nlay();
    const int ngpt = optical_props->get_ngpt();

    Array_gpu<TF,2> sfc_alb_dir_gpt({ncol, ngpt});
    Array_gpu<TF,2> sfc_alb_dif_gpt({ncol, ngpt});

    expand_and_transpose(optical_props, sfc_alb_dir, sfc_alb_dir_gpt);
    expand_and_transpose(optical_props, sfc_alb_dif, sfc_alb_dif_gpt);

    // Upper boundary condition. At this stage, flux_dn contains the diffuse radiation only.
    rte_kernel_launcher_cuda::apply_BC(ncol, nlay, ngpt, top_at_1, inc_flux_dir, mu0, gpt_flux_dir);
    if (inc_flux_dif.size() == 0)
        rte_kernel_launcher_cuda::apply_BC(ncol, nlay, ngpt, top_at_1, gpt_flux_dn);
    else
        rte_kernel_launcher_cuda::apply_BC(ncol, nlay, ngpt, top_at_1, inc_flux_dif, gpt_flux_dn);

    // Run the radiative transfer solver
    // CvH: only two-stream solutions, I skipped the sw_solver_noscat.
    rte_kernel_launcher_cuda::sw_solver_2stream(
            ncol, nlay, ngpt, top_at_1,
            optical_props->get_tau(),
            optical_props->get_ssa(),
            optical_props->get_g  (),
            mu0,
            sfc_alb_dir_gpt, sfc_alb_dif_gpt,
            gpt_flux_up, gpt_flux_dn, gpt_flux_dir);

    // CvH: The original fortran code had a call to the reduce here.
    // fluxes->reduce(gpt_flux_up, gpt_flux_dn, gpt_flux_dir, optical_props, top_at_1);
}

template<typename TF>
void Rte_sw_gpu<TF>::expand_and_transpose(
        const std::unique_ptr<Optical_props_arry_gpu<TF>>& ops,
        const Array_gpu<TF,2> arr_in,
        Array_gpu<TF,2>& arr_out)
{
    const int ncol = arr_in.dim(2);
    const int nbnd = ops->get_nband();
    const int block_col = 16;
    const int block_bnd = 14;

    const int grid_col = ncol/block_col + (ncol%block_col > 0);
    const int grid_bnd = nbnd/block_bnd + (nbnd%block_bnd > 0);

    dim3 grid_gpu(grid_col, grid_bnd);
    dim3 block_gpu(block_col, block_bnd);

    Array_gpu<int,2> limits = ops->get_band_lims_gpoint_gpu();

    // Array_gpu<int,2> limits(limitsc);
    expand_and_transpose_kernel<<<grid_gpu, block_gpu>>>(
        ncol, nbnd, limits.ptr(), arr_out.ptr(), arr_in.ptr());
}

#ifdef FLOAT_SINGLE_RRTMGP
template class Rte_sw_gpu<float>;
#else
template class Rte_sw_gpu<double>;
#endif
