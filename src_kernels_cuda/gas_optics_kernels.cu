#include "hip/hip_runtime.h"
#include "Types.h"

template<typename TF>__device__
TF interpolate1D(const TF val,
                 const TF offset,
                 const TF delta,
                 const int len,
                 const TF* __restrict__ table)
{
    TF val0 = (val - offset)/delta;
    TF frac = val0 - int(val0);
    int idx = min(len-1, max(1, int(val0)+1));
    return table[idx-1] + frac * (table[idx] - table[idx-1]);
}


template<typename TF>__device__
void interpolate2D_byflav_kernel(const TF* __restrict__ fminor,
                                 const TF* __restrict__ kin,
                                 const int gpt_start, const int gpt_end,
                                 TF* __restrict__ k,
                                 const int* __restrict__ jeta,
                                 const int jtemp,
                                 const int ngpt,
                                 const int neta)
{
    const int band_gpt = gpt_end-gpt_start;
    const int j0 = jeta[0];
    const int j1 = jeta[1];
    for (int igpt=0; igpt<band_gpt; ++igpt)
    {
        k[igpt] = fminor[0] * kin[igpt + (j0-1)*ngpt + (jtemp-1)*neta*ngpt] +
                  fminor[1] * kin[igpt +  j0   *ngpt + (jtemp-1)*neta*ngpt] +
                  fminor[2] * kin[igpt + (j1-1)*ngpt + jtemp    *neta*ngpt] +
                  fminor[3] * kin[igpt +  j1   *ngpt + jtemp    *neta*ngpt];
    }
}


template<typename TF>__device__
void interpolate3D_byflav_kernel(const TF* __restrict__ scaling,
                                 const TF* __restrict__ fmajor,
                                 const TF* __restrict__ k,
                                 const int gpt_start, const int gpt_end,
                                 const int* __restrict__ jeta,
                                 const int jtemp,
                                 const int jpress,
                                 const int ngpt,
                                 const int neta,
                                 const int npress,
                                 TF* __restrict__ tau_major)
{
    const int band_gpt = gpt_end-gpt_start;
    const int j0 = jeta[0];
    const int j1 = jeta[1];
    for (int igpt=0; igpt<band_gpt; ++igpt)
    {
        tau_major[igpt] = scaling[0]*
                          (fmajor[0] * k[igpt + (j0-1)*ngpt + (jpress-1)*neta*ngpt + (jtemp-1)*neta*ngpt*npress] +
                           fmajor[1] * k[igpt +  j0   *ngpt + (jpress-1)*neta*ngpt + (jtemp-1)*neta*ngpt*npress] +
                           fmajor[2] * k[igpt + (j0-1)*ngpt + jpress*neta*ngpt     + (jtemp-1)*neta*ngpt*npress] +
                           fmajor[3] * k[igpt +  j0   *ngpt + jpress*neta*ngpt     + (jtemp-1)*neta*ngpt*npress])
                        + scaling[1]*
                          (fmajor[4] * k[igpt + (j1-1)*ngpt + (jpress-1)*neta*ngpt + jtemp*neta*ngpt*npress] +
                           fmajor[5] * k[igpt +  j1   *ngpt + (jpress-1)*neta*ngpt + jtemp*neta*ngpt*npress] +
                           fmajor[6] * k[igpt + (j1-1)*ngpt + jpress*neta*ngpt     + jtemp*neta*ngpt*npress] +
                           fmajor[7] * k[igpt +  j1   *ngpt + jpress*neta*ngpt     + jtemp*neta*ngpt*npress]);
    }
}


template<typename TF>__global__
void reorder12x21_kernel(
        const int ni, const int nj,
        const TF* __restrict__ arr_in, TF* __restrict__ arr_out)
{
    const int ii = blockIdx.x*blockDim.x + threadIdx.x;
    const int ij = blockIdx.y*blockDim.y + threadIdx.y;
    if ( (ii < ni) && (ij < nj) )
    {
        const int idx_out = ii + ij*ni;
        const int idx_in = ij + ii*nj;
        arr_out[idx_out] = arr_in[idx_in];
    }
}


template<typename TF>__global__
void reorder123x321_kernel(
        const int ni, const int nj, const int nk,
        const TF* __restrict__ arr_in, TF* __restrict__ arr_out)
{
    const int ii = blockIdx.x*blockDim.x + threadIdx.x;
    const int ij = blockIdx.y*blockDim.y + threadIdx.y;
    const int ik = blockIdx.z*blockDim.z + threadIdx.z;
    if ( (ii < ni) && (ij < nj) && (ik < nk))
    {
        const int idx_out = ii + ij*ni + ik*nj*ni;
        const int idx_in = ik + ij*nk + ii*nj*nk;
        arr_out[idx_out] = arr_in[idx_in];
    }
}


template<typename TF>__global__
void zero_array_kernel(
        const int ni, const int nj, const int nk,
        TF* __restrict__ arr)
{
    const int ii = blockIdx.x*blockDim.x + threadIdx.x;
    const int ij = blockIdx.y*blockDim.y + threadIdx.y;
    const int ik = blockIdx.z*blockDim.z + threadIdx.z;
    if ( (ii < ni) && (ij < nj) && (ik < nk))
    {
        const int idx = ii + ij*ni + ik*nj*ni;
        arr[idx] = TF(0.);
    }
}


template<typename TF>__global__
void Planck_source_kernel(
        const int ncol, const int nlay, const int nband, const int ngpt,
        const int nflav, const int neta, const int npres, const int ntemp,
        const int nPlanckTemp,
        const TF* __restrict__ tlay, const TF* __restrict__ tlev,
        const TF* __restrict__ tsfc,
        const int sfc_lay,
        const TF* __restrict__ fmajor, const int* __restrict__ jeta,
        const BOOL_TYPE* __restrict__ tropo, const int* __restrict__ jtemp,
        const int* __restrict__ jpress, const int* __restrict__ gpoint_bands,
        const int* __restrict__ band_lims_gpt, const TF* __restrict__ pfracin,
        const TF temp_ref_min, const TF totplnk_delta,
        const TF* __restrict__ totplnk, const int* __restrict__ gpoint_flavor,
        const TF* __restrict__ ones, const TF delta_Tsurf,
        TF* __restrict__ sfc_src, TF* __restrict__ lay_src,
        TF* __restrict__ lev_src_inc, TF* __restrict__ lev_src_dec,
        TF* __restrict__ sfc_src_jac, TF* __restrict__ pfrac)
{
    const int igpt = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
    const int icol = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
    {

        //retrieve the band (naive implementation)//
        int ibnd = -1;
        int lbnd =  0;
        while(ibnd == -1)
	{
	  if( (igpt >= (band_lims_gpt[2 * ibnd] - 1)) && (igpt < band_lims_gpt[2 * ibnd + 1]) )
	  {
	    ibnd = lbnd;
	  }
	  lbnd = lbnd +1;
	  //if(lbnd > nband) return;
	}

        const int idx_collay = icol + ilay * ncol;
        const int itropo = !tropo[idx_collay];
        const int gpt_start = band_lims_gpt[2 * ibnd] - 1;
	//        const int gpt_end = band_lims_gpt[2 * ibnd + 1];
        const int iflav = gpoint_flavor[itropo + 2 * gpt_start] - 1;
        const int idx_fcl3 = 2 * 2 * 2* (iflav + icol * nflav + ilay * ncol * nflav);
        const int idx_fcl1 = 2 * (iflav + icol * nflav + ilay * ncol * nflav);
        //const int idx_tau = gpt_start + ilay * ngpt + icol * nlay * ngpt;
	const int idx_in = igpt + ilay * ngpt + icol * nlay * ngpt;

        //major gases//
        // interpolate3D_byflav_kernel(ones, &fmajor[idx_fcl3],
        //                             &pfracin[gpt_start], gpt_start, gpt_end,
        //                             &jeta[idx_fcl1], jtemp[idx_collay],
        //                             jpress[idx_collay]+itropo, ngpt, neta, npres+1,
        //                             &pfrac[idx_tau]);

	interpolate3D_byflav_kernel(ones, &fmajor[idx_fcl3],
                                    &pfracin[igpt], igpt, igpt+1,
                                    &jeta[idx_fcl1], jtemp[idx_collay],
                                    jpress[idx_collay]+itropo, ngpt, neta, npres+1,
                                    &pfrac[idx_in]);

        // compute surface source irradiances
        if (ilay == sfc_lay - 1) // Subtract one to correct for fortran indexing.
        {
            const TF planck_function_sfc1 = interpolate1D(tsfc[icol],               temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);
            const TF planck_function_sfc2 = interpolate1D(tsfc[icol] + delta_Tsurf, temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);

	    const int idx_in  = igpt + ilay*ngpt + icol*nlay*ngpt;
	    const int idx_out = igpt + icol*ngpt;
	    sfc_src[idx_out] = pfrac[idx_in] * planck_function_sfc1;
	    sfc_src_jac[idx_out] = pfrac[idx_in] * (planck_function_sfc2 - planck_function_sfc1);
        }

        // compute layer source irradiances.
        const int idx_tmp = icol + ilay*ncol;
        const TF planck_function_lay = interpolate1D(tlay[idx_tmp], temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);

	const int idx_inout  = igpt + ilay*ngpt + icol*nlay*ngpt;
	lay_src[idx_inout] = pfrac[idx_inout] * planck_function_lay;


        // compute level source irradiances.
        const int idx_tmp1 = icol + (ilay+1)*ncol;
        const int idx_tmp2 = icol + ilay*ncol;
        const TF planck_function_lev1 = interpolate1D(tlev[idx_tmp1], temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);
        const TF planck_function_lev2 = interpolate1D(tlev[idx_tmp2], temp_ref_min, totplnk_delta, nPlanckTemp, &totplnk[ibnd * nPlanckTemp]);

	const int idx_inout  = igpt + ilay*ngpt + icol*nlay*ngpt;
	lev_src_inc[idx_inout] = pfrac[idx_inout] * planck_function_lev1;
	lev_src_dec[idx_inout] = pfrac[idx_inout] * planck_function_lev2;
    }
}


template<typename TF>__global__
void interpolation_kernel(
        const int ncol, const int nlay, const int ngas, const int nflav,
        const int neta, const int npres, const int ntemp, const TF tmin,
        const int* __restrict__ flavor,
        const TF* __restrict__ press_ref_log,
        const TF* __restrict__ temp_ref,
        TF press_ref_log_delta,
        TF temp_ref_min,
        TF temp_ref_delta,
        TF press_ref_trop_log,
        const TF* __restrict__ vmr_ref,
        const TF* __restrict__ play,
        const TF* __restrict__ tlay,
        TF* __restrict__ col_gas,
        int* __restrict__ jtemp,
        TF* __restrict__ fmajor, TF* __restrict__ fminor,
        TF* __restrict__ col_mix,
        BOOL_TYPE* __restrict__ tropo,
        int* __restrict__ jeta,
        int* __restrict__ jpress)
{
    const int ilay = blockIdx.x*blockDim.x + threadIdx.x;
    const int icol = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (ilay < nlay) )
    {
        const int idx = icol + ilay*ncol;

        jtemp[idx] = int((tlay[idx] - (temp_ref_min-temp_ref_delta)) / temp_ref_delta);
        jtemp[idx] = min(ntemp-1, max(1, jtemp[idx]));
        const TF ftemp = (tlay[idx] - temp_ref[jtemp[idx]-1]) / temp_ref_delta;

        const TF locpress = TF(1.) + (log(play[idx]) - press_ref_log[0]) / press_ref_log_delta;
        jpress[idx] = min(npres-1, max(1, int(locpress)));
        const TF fpress = locpress - TF(jpress[idx]);

        tropo[idx] = log(play[idx]) > press_ref_trop_log;
        const int itropo = !tropo[idx];

        for (int iflav=0; iflav<nflav; ++iflav)
        {
            const int gas1 = flavor[2*iflav];
            const int gas2 = flavor[2*iflav+1];
            for (int itemp=0; itemp<2; ++itemp)
            {
                const int vmr_base_idx = itropo + (jtemp[idx]+itemp-1) * (ngas+1) * 2;
                const int colmix_idx = itemp + 2*(iflav + nflav*icol + nflav*ncol*ilay);
                const int colgas1_idx = icol + ilay*ncol + gas1*nlay*ncol;
                const int colgas2_idx = icol + ilay*ncol + gas2*nlay*ncol;
                TF eta;
                const TF ratio_eta_half = vmr_ref[vmr_base_idx + 2 * gas1] /
                                          vmr_ref[vmr_base_idx + 2 * gas2];
                col_mix[colmix_idx] = col_gas[colgas1_idx] + ratio_eta_half * col_gas[colgas2_idx];
                if (col_mix[colmix_idx] > TF(2.)*tmin)
                {
                    eta = col_gas[colgas1_idx] / col_mix[colmix_idx];
                } else
                {
                    eta = TF(0.5);
                }
                const TF loceta = eta * TF(neta-1);
                jeta[colmix_idx] = min(int(loceta)+1, neta-1);
                const TF feta = fmod(loceta, TF(1.));
                const TF ftemp_term  = TF(1-itemp) + TF(2*itemp-1)*ftemp;
                // compute interpolation fractions needed for minot species
                const int fminor_idx = 2*(itemp + 2*(iflav + icol*nflav + ilay*ncol*nflav));
                fminor[fminor_idx] = (TF(1.0)-feta) * ftemp_term;
                fminor[fminor_idx+1] = feta * ftemp_term;
                // compute interpolation fractions needed for major species
                const int fmajor_idx = 2*2*(itemp + 2*(iflav + icol*nflav + ilay*ncol*nflav));
                fmajor[fmajor_idx] = (TF(1.0)-fpress) * fminor[fminor_idx];
                fmajor[fmajor_idx+1] = (TF(1.0)-fpress) * fminor[fminor_idx+1];
                fmajor[fmajor_idx+2] = fpress * fminor[fminor_idx];
                fmajor[fmajor_idx+3] = fpress * fminor[fminor_idx+1];

            }
        }
    }
}


template<typename TF>__global__
void compute_tau_major_absorption_kernel(
        const int ncol, const int nlay, const int nband, const int ngpt,
        const int nflav, const int neta, const int npres, const int ntemp,
        const int* __restrict__ gpoint_flavor,
        const int* __restrict__ band_lims_gpt,
        const TF* __restrict__ kmajor,
        const TF* __restrict__ col_mix, const TF* __restrict__ fmajor,
        const int* __restrict__ jeta, const BOOL_TYPE* __restrict__ tropo,
        const int* __restrict__ jtemp, const int* __restrict__ jpress,
        TF* __restrict__ tau, TF* __restrict__ tau_major)
{
    // Fetch the three coordinates.
    const int ibnd = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
    const int icol = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (ibnd < nband) ) {
        const int idx_collay = icol + ilay * ncol;
        const int itropo = !tropo[idx_collay];
        const int gpt_start = band_lims_gpt[2 * ibnd] - 1;
        const int gpt_end = band_lims_gpt[2 * ibnd + 1];
        const int iflav = gpoint_flavor[itropo + 2 * gpt_start] - 1;
        const int idx_fcl3 = 2 * 2 * 2* (iflav + icol * nflav + ilay * ncol * nflav);
        const int idx_fcl1 = 2 * (iflav + icol * nflav + ilay * ncol * nflav);
        const int idx_tau = gpt_start + ilay * ngpt + icol * nlay * ngpt;

        //major gases//
        interpolate3D_byflav_kernel(&col_mix[idx_fcl1], &fmajor[idx_fcl3],
                                    &kmajor[gpt_start], gpt_start, gpt_end,
                                    &jeta[idx_fcl1], jtemp[idx_collay],
                                    jpress[idx_collay]+itropo, ngpt, neta, npres+1,
                                    &tau_major[idx_tau]);

        for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
        {
            const int idx_out = igpt + ilay*ngpt + icol*nlay*ngpt;
            tau[idx_out] += tau_major[idx_out];
        }
    }
}


template<typename TF>__global__
void compute_tau_minor_absorption_kernel(
        const int ncol, const int nlay, const int ngpt,
        const int ngas, const int nflav, const int ntemp, const int neta,
        const int nscale_lower,
        const int nscale_upper,
        const int nminor_lower,
        const int nminor_upper,
        const int nminork_lower,
        const int nminork_upper,
        const int idx_h2o,
        const int* __restrict__ gpoint_flavor,
        const TF* __restrict__ kminor_lower,
        const TF* __restrict__ kminor_upper,
        const int* __restrict__ minor_limits_gpt_lower,
        const int* __restrict__ minor_limits_gpt_upper,
        const BOOL_TYPE* __restrict__ minor_scales_with_density_lower,
        const BOOL_TYPE* __restrict__ minor_scales_with_density_upper,
        const BOOL_TYPE* __restrict__ scale_by_complement_lower,
        const BOOL_TYPE* __restrict__ scale_by_complement_upper,
        const int* __restrict__ idx_minor_lower,
        const int* __restrict__ idx_minor_upper,
        const int* __restrict__ idx_minor_scaling_lower,
        const int* __restrict__ idx_minor_scaling_upper,
        const int* __restrict__ kminor_start_lower,
        const int* __restrict__ kminor_start_upper,
        const TF* __restrict__ play,
        const TF* __restrict__ tlay,
        const TF* __restrict__ col_gas,
        const TF* __restrict__ fminor,
        const int* __restrict__ jeta,
        const int* __restrict__ jtemp,
        const BOOL_TYPE* __restrict__ tropo,
        TF* __restrict__ tau,
        TF* __restrict__ tau_minor)
{
    // Fetch the three coordinates.
    const int ilay = blockIdx.x * blockDim.x + threadIdx.x;
    const int icol = blockIdx.y * blockDim.y + threadIdx.y;
    const TF PaTohPa = 0.01;
    const int ncl = ncol * nlay;
    if ((icol < ncol) && (ilay < nlay))
    {
        //kernel implementation
        const int idx_collay = icol + ilay * ncol;
        const int idx_collaywv = icol + ilay * ncol + idx_h2o * ncl;

        if (tropo[idx_collay] == 1)
        {
            for (int imnr = 0; imnr < nscale_lower; ++imnr)
            {
                TF scaling = col_gas[idx_collay + idx_minor_lower[imnr] * ncl];
                if (minor_scales_with_density_lower[imnr])
                {
                    scaling *= PaTohPa * play[idx_collay] / tlay[idx_collay];
                    if (idx_minor_scaling_lower[imnr] > 0)
                    {
                        TF vmr_fact = TF(1.) / col_gas[idx_collay];
                        TF dry_fact = TF(1.) / (TF(1.) + col_gas[idx_collaywv] * vmr_fact);
                        if (scale_by_complement_lower[imnr])
                        {
                            scaling *= (TF(1.) - col_gas[idx_collay + idx_minor_scaling_lower[imnr] * ncl] * vmr_fact * dry_fact);
                        }
                        else
                        {
                            scaling *= col_gas[idx_collay + idx_minor_scaling_lower[imnr] * ncl] * vmr_fact * dry_fact;
                        }
                    }
                }
                const int gpt_start = minor_limits_gpt_lower[2*imnr]-1;
                const int gpt_end = minor_limits_gpt_lower[2*imnr+1];
                const int iflav = gpoint_flavor[2*gpt_start]-1;
                const int idx_fcl2 = 2 * 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                const int idx_fcl1 = 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                const int idx_tau = gpt_start + ilay*ngpt + icol*nlay*ngpt;

                interpolate2D_byflav_kernel(&fminor[idx_fcl2], &kminor_lower[kminor_start_lower[imnr]-1],
                                            kminor_start_lower[imnr]-1, kminor_start_lower[imnr]-1 + (gpt_end - gpt_start),
                                            &tau_minor[idx_tau], &jeta[idx_fcl1],
                                            jtemp[idx_collay], nminork_lower, neta);

                for (int igpt = gpt_start; igpt < gpt_end; ++igpt)
                {
                    const int idx_out = igpt + ilay * ngpt + icol * nlay * ngpt;
                    tau[idx_out] += tau_minor[idx_out] * scaling;
                }
            }
        }
        else
        {
            for (int imnr = 0; imnr < nscale_upper; ++imnr)
            {
                TF scaling = col_gas[idx_collay + idx_minor_upper[imnr] * ncl];
                if (minor_scales_with_density_upper[imnr])
                {
                    scaling *= PaTohPa * play[idx_collay] / tlay[idx_collay];
                    if (idx_minor_scaling_upper[imnr] > 0)
                    {
                        TF vmr_fact = TF(1.) / col_gas[idx_collay];
                        TF dry_fact = TF(1.) / (TF(1.) + col_gas[idx_collaywv] * vmr_fact);
                        if (scale_by_complement_upper[imnr])
                        {
                            scaling *= (TF(1.) - col_gas[idx_collay + idx_minor_scaling_upper[imnr] * ncl] * vmr_fact * dry_fact);
                        }
                        else
                        {
                            scaling *= col_gas[idx_collay + idx_minor_scaling_upper[imnr] * ncl] * vmr_fact * dry_fact;
                        }
                    }
                }
                const int gpt_start = minor_limits_gpt_upper[2*imnr]-1;
                const int gpt_end = minor_limits_gpt_upper[2*imnr+1];
                const int iflav = gpoint_flavor[2*gpt_start+1]-1;
                const int idx_fcl2 = 2 * 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                const int idx_fcl1 = 2 * (iflav + icol * nflav + ilay * ncol * nflav);
                const int idx_tau = gpt_start + ilay*ngpt + icol*nlay*ngpt;

                interpolate2D_byflav_kernel(&fminor[idx_fcl2], &kminor_upper[kminor_start_upper[imnr]-1],
                                            kminor_start_upper[imnr]-1, kminor_start_upper[imnr]-1 + (gpt_end - gpt_start),
                                            &tau_minor[idx_tau], &jeta[idx_fcl1],
                                            jtemp[idx_collay], nminork_upper, neta);

                for (int igpt = gpt_start; igpt < gpt_end; ++igpt)
                {
                    const int idx_out = igpt + ilay * ngpt + icol * nlay * ngpt;
                    tau[idx_out] += tau_minor[idx_out] * scaling;
                }
            }
        }
    }
}


template<typename TF>__global__
void compute_tau_rayleigh_kernel(
        const int ncol, const int nlay, const int nbnd, const int ngpt,
        const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
        const int* __restrict__ gpoint_flavor,
        const int* __restrict__ band_lims_gpt,
        const TF* __restrict__ krayl,
        int idx_h2o, const TF* __restrict__ col_dry, const TF* __restrict__ col_gas,
        const TF* __restrict__ fminor, const int* __restrict__ jeta,
        const BOOL_TYPE* __restrict__ tropo, const int* __restrict__ jtemp,
        TF* __restrict__ tau_rayleigh, TF* __restrict__ k)
{
    // Fetch the three coordinates.
    const int ibnd = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
    const int icol = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (ibnd < nbnd) )
    {
        //kernel implementation
        const int idx_collay = icol + ilay*ncol;
        const int idx_collaywv = icol + ilay*ncol + idx_h2o*nlay*ncol;
        const int itropo = !tropo[idx_collay];
        const int gpt_start = band_lims_gpt[2*ibnd]-1;
        const int gpt_end = band_lims_gpt[2*ibnd+1];
        const int iflav = gpoint_flavor[itropo+2*gpt_start]-1;
        const int idx_fcl2 = 2*2*(iflav + icol*nflav + ilay*ncol*nflav);
        const int idx_fcl1   = 2*(iflav + icol*nflav + ilay*ncol*nflav);
        const int idx_krayl  = gpt_start + ngpt*neta*ntemp*itropo;
        const int idx_k = gpt_start + ilay*ngpt + icol*nlay*ngpt;
        interpolate2D_byflav_kernel(&fminor[idx_fcl2],
                                    &krayl[idx_krayl],
                                    gpt_start, gpt_end, &k[idx_k],
                                    &jeta[idx_fcl1],
                                    jtemp[idx_collay],
                                    ngpt, neta);

        for (int igpt=gpt_start; igpt<gpt_end; ++igpt)
        {
            const int idx_out = igpt + ilay*ngpt + icol*nlay*ngpt;
            tau_rayleigh[idx_out] = k[idx_k+igpt-gpt_start]*(col_gas[idx_collaywv]+col_dry[idx_collay]);
        }
    }
}


template<typename TF>__global__
void combine_and_reorder_2str_kernel(
        const int ncol, const int nlay, const int ngpt, const TF tmin,
        const TF* __restrict__ tau_abs, const TF* __restrict__ tau_rayleigh,
        TF* __restrict__ tau, TF* __restrict__ ssa, TF* __restrict__ g)
{
    // Fetch the three coordinates.
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int igpt = blockIdx.y*blockDim.y + threadIdx.y;
    const int ilay = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
    {
        const int idx_in  = igpt + ilay*ngpt + icol*(ngpt*nlay);
        const int idx_out = icol + ilay*ncol + igpt*(ncol*nlay);

        const TF tau_tot = tau_abs[idx_in] + tau_rayleigh[idx_in];
        tau[idx_out] = tau_tot;
        g  [idx_out] = TF(0.);
        if (tau_tot>(TF(2.)*tmin))
            ssa[idx_out] = tau_rayleigh[idx_in]/tau_tot;
        else
            ssa[idx_out] = TF(0.);
    }
}

