#include "hip/hip_runtime.h"
#include <chrono>

#include "rte_kernel_launcher_cuda.h"
#include "tools_gpu.h"
#include "Array.h"
#include <iomanip>

namespace
{
    template<typename TF>__device__
    void lw_source_noscat_kernel(const int icol, const int igpt, const int ncol, const int nlay, const int ngpt, const TF eps,
                                 const TF* __restrict__ lay_source, const TF* __restrict__ lev_source_up, const TF* __restrict__ lev_source_dn,
                                 const TF* __restrict__ tau, const TF* __restrict__ trans, TF* __restrict__ source_dn, TF* __restrict__ source_up)
    {
        const TF tau_thres = sqrt(eps);
        for (int ilay=0; ilay<nlay; ++ilay)
        {
            const int idx = icol + ilay*ncol + igpt*ncol*nlay;
            const TF fact = (tau[idx]>tau_thres) ? (TF(1.) - trans[idx]) / tau[idx] - trans[idx] : tau[idx] * (TF(.5) - TF(1.)/TF(3.)*tau[idx]);
            source_dn[idx] = (TF(1.) - trans[idx]) * lev_source_dn[idx] + TF(2.) * fact * (lay_source[idx]-lev_source_dn[idx]);
            source_up[idx] = (TF(1.) - trans[idx]) * lev_source_up[idx] + TF(2.) * fact * (lay_source[idx]-lev_source_up[idx]);
        }
    }

    template<typename TF>__device__
    void lw_transport_noscat_kernel(const int icol, const int igpt, const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1,
                                 const TF* __restrict__ tau, const TF* __restrict__ trans, const TF* __restrict__ sfc_albedo,
                                 const TF* __restrict__ source_dn, const TF* __restrict__ source_up, const TF* __restrict__ source_sfc,
                                 TF* __restrict__ radn_up, TF* __restrict__ radn_dn, const TF* __restrict__ source_sfc_jac, TF* __restrict__ radn_up_jac)
    {
        if (top_at_1)
        {
            for (int ilev=1; ilev<(nlay+1); ++ilev)
            {
                const int idx1 = icol + ilev*ncol + igpt*ncol*(nlay+1);
                const int idx2 = icol + (ilev-1)*ncol + igpt*ncol*(nlay+1);
                const int idx3 = icol + (ilev-1)*ncol + igpt*ncol*nlay;
                radn_dn[idx1] = trans[idx3] * radn_dn[idx2] + source_dn[idx3];
            }

            const int idx_bot = icol + nlay*ncol + igpt*ncol*(nlay+1);
            const int idx2d = icol + igpt*ncol;
            radn_up[idx_bot] = radn_dn[idx_bot] * sfc_albedo[idx2d] + source_sfc[idx2d];
            radn_up_jac[idx_bot] = source_sfc_jac[idx2d];

            for (int ilev=nlay-1; ilev>=0; --ilev)
            {
                const int idx1 = icol + ilev*ncol + igpt*ncol*(nlay+1);
                const int idx2 = icol + (ilev+1)*ncol + igpt*ncol*(nlay+1);
                const int idx3 = icol + ilev*ncol + igpt*ncol*nlay;
                radn_up[idx1] = trans[idx3] * radn_up[idx2] + source_up[idx3];
                radn_up_jac[idx1] = trans[idx3] * radn_up_jac[idx2];
            }
        }
        else
        {
            for (int ilev=(nlay-1); ilev>=0; --ilev)
            {
                const int idx1 = icol + ilev*ncol + igpt*ncol*(nlay+1);
                const int idx2 = icol + (ilev+1)*ncol + igpt*ncol*(nlay+1);
                const int idx3 = icol + ilev*ncol + igpt*ncol*nlay;
                radn_dn[idx1] = trans[idx3] * radn_dn[idx2] + source_dn[idx3];
            }

            const int idx_bot = icol + igpt*ncol*(nlay+1);
            const int idx2d = icol + igpt*ncol;
            radn_up[idx_bot] = radn_dn[idx_bot] * sfc_albedo[idx2d] + source_sfc[idx2d];
            radn_up_jac[idx_bot] = source_sfc_jac[idx2d];

            for (int ilev=1; ilev<(nlay+1); ++ilev)
            {
                const int idx1 = icol + ilev*ncol + igpt*ncol*(nlay+1);
                const int idx2 = icol + (ilev-1)*ncol + igpt*ncol*(nlay+1);
                const int idx3 = icol + (ilev-1)*ncol + igpt*ncol*nlay;
                radn_up[idx1] = trans[idx3] * radn_up[idx2] + source_up[idx3];
                radn_up_jac[idx1] = trans[idx3] * radn_up_jac[idx2];;
            }
        }

    }

    template<typename TF>__device__
    void lw_solver_noscat_kernel(const int icol, const int igpt, const int ncol, const int nlay, const int ngpt, const TF eps, const BOOL_TYPE top_at_1,
                                 const TF D, const TF weight, const TF* __restrict__ tau, const TF* __restrict__ lay_source,
                                 const TF* __restrict__ lev_source_inc, const TF* __restrict__ lev_source_dec, const TF* __restrict__ sfc_emis,
                                 const TF* __restrict__ sfc_src, TF* __restrict__ radn_up, TF* __restrict__ radn_dn,
                                 const TF* __restrict__ sfc_src_jac, TF* __restrict__ radn_up_jac, TF* __restrict__ tau_loc,
                                 TF* __restrict__ trans, TF* __restrict__ source_dn, TF* __restrict__ source_up,
                                 TF* __restrict__ source_sfc, TF* __restrict__ sfc_albedo, TF* __restrict__ source_sfc_jac)
    {
        const TF pi = acos(TF(-1.));
        const TF* lev_source_up;
        const TF* lev_source_dn;
        int top_level;
        if (top_at_1)
        {
            top_level = 0;
            lev_source_up = lev_source_dec;
            lev_source_dn = lev_source_inc;
        }
        else
        {
            top_level = nlay;
            lev_source_up = lev_source_inc;
            lev_source_dn = lev_source_dec;
        }
        const int idx_top = icol + top_level*ncol + igpt*ncol*(nlay+1);
        radn_dn[idx_top] = radn_dn[idx_top] / (TF(2.) * pi * weight);

        const int idx2d = icol + igpt*ncol;

        for (int ilay=0; ilay<nlay; ++ilay)
        {
            const int idx3d = icol + ilay*ncol + igpt*ncol*nlay;
            tau_loc[idx3d] = tau[idx3d] * D;
            trans[idx3d]   = exp(-tau_loc[idx3d]);
        }

        lw_source_noscat_kernel(icol, igpt, ncol, nlay, ngpt, eps, lay_source, lev_source_up, lev_source_dn,
                         tau_loc, trans, source_dn, source_up);

        sfc_albedo[idx2d] = TF(1.) - sfc_emis[idx2d];
        source_sfc[idx2d] = sfc_emis[idx2d] * sfc_src[idx2d];
        source_sfc_jac[idx2d] = sfc_emis[idx2d] * sfc_src_jac[idx2d];

        lw_transport_noscat_kernel(icol, igpt, ncol, nlay, ngpt, top_at_1, tau, trans, sfc_albedo, source_dn,
                                   source_up, source_sfc, radn_up, radn_dn, source_sfc_jac, radn_up_jac);

        for (int ilev=0; ilev<(nlay+1); ++ilev)
        {
            const int idx = icol + ilev*ncol + igpt*ncol*(nlay+1);
            radn_up[idx] *= TF(2.) * pi * weight;
            radn_dn[idx] *= TF(2.) * pi * weight;
            radn_up_jac[idx] *= TF(2.) * pi * weight;
        }
    }

    template<typename TF>__device__
    void sw_adding_kernel(const int icol, const int igpt,
                          const int ncol, const int nlay, const BOOL_TYPE top_at_1,
                          const TF* __restrict__ sfc_alb_dif, const TF* __restrict__ r_dif, const TF* __restrict__ t_dif,
                          const TF* __restrict__ source_dn, const TF* __restrict__ source_up, const TF* __restrict__ source_sfc,
                          TF* __restrict__ flux_up, TF* __restrict__ flux_dn, const TF* __restrict__ flux_dir,
                          TF* __restrict__ albedo, TF* __restrict__ src, TF* __restrict__ denom)
    {
        if (top_at_1)
        {
            const int sfc_idx_3d = icol + nlay*ncol + igpt*(nlay+1)*ncol;
            const int sfc_idx_2d = icol + igpt*ncol;
            albedo[sfc_idx_3d] = sfc_alb_dif[sfc_idx_2d];
            src[sfc_idx_3d] = source_sfc[sfc_idx_2d];

            for (int ilay=nlay-1; ilay >= 0; --ilay)
            {
                const int lay_idx  = icol + ilay*ncol + igpt*ncol*nlay;
                const int lev_idx1 = icol + ilay*ncol + igpt*ncol*(nlay+1);
                const int lev_idx2 = icol + (ilay+1)*ncol + igpt*ncol*(nlay+1);
                denom[lay_idx] = TF(1.)/(TF(1.) - r_dif[lay_idx] * albedo[lev_idx2]);
                albedo[lev_idx1] = r_dif[lay_idx] + t_dif[lay_idx] * t_dif[lay_idx]
                                                  * albedo[lev_idx2] * denom[lay_idx];
                src[lev_idx1] = source_up[lay_idx] + t_dif[lay_idx] * denom[lay_idx] *
                                (src[lev_idx2] + albedo[lev_idx2] * source_dn[lay_idx]);
            }
            const int top_idx = icol + igpt*(nlay+1)*ncol;
            flux_up[top_idx] = flux_dn[top_idx]*albedo[top_idx] + src[top_idx];

            for (int ilay=1; ilay < (nlay+1); ++ilay)
            {
                const int lev_idx1 = icol + ilay*ncol + igpt*(nlay+1)*ncol;
                const int lev_idx2 = icol + (ilay-1)*ncol + igpt*(nlay+1)*ncol;
                const int lay_idx = icol + (ilay-1)*ncol + igpt*(nlay)*ncol;
                flux_dn[lev_idx1] = (t_dif[lay_idx]*flux_dn[lev_idx2] +
                                     r_dif[lay_idx]*src[lev_idx1] +
                                     source_dn[lay_idx]) * denom[lay_idx];
                flux_up[lev_idx1] = flux_dn[lev_idx1] * albedo[lev_idx1] + src[lev_idx1];
            }

            for (int ilay=0; ilay < (nlay+1); ++ilay)
            {
                const int idx = icol + ilay*ncol + igpt*(nlay+1)*ncol;
                flux_dn[idx] += flux_dir[idx];
            }
        }
        else
        {
            const int sfc_idx_3d = icol + igpt*(nlay+1)*ncol;
            const int sfc_idx_2d = icol + igpt*ncol;
            albedo[sfc_idx_3d] = sfc_alb_dif[sfc_idx_2d];
            src[sfc_idx_3d] = source_sfc[sfc_idx_2d];

            for (int ilay=0; ilay<nlay; ++ilay)
            {
                const int lay_idx  = icol + ilay*ncol + igpt*ncol*nlay;
                const int lev_idx1 = icol + ilay*ncol + igpt*ncol*(nlay+1);
                const int lev_idx2 = icol + (ilay+1)*ncol + igpt*ncol*(nlay+1);
                denom[lay_idx] = TF(1.)/(TF(1.) - r_dif[lay_idx] * albedo[lev_idx1]);
                albedo[lev_idx2] = r_dif[lay_idx] + (t_dif[lay_idx] * t_dif[lay_idx] *
                                                     albedo[lev_idx1] * denom[lay_idx]);
                src[lev_idx2] = source_up[lay_idx] + t_dif[lay_idx]*denom[lay_idx]*
                                                     (src[lev_idx1]+albedo[lev_idx1]*source_dn[lay_idx]);
            }
            const int top_idx = icol + nlay*ncol + igpt*(nlay+1)*ncol;
            flux_up[top_idx] = flux_dn[top_idx] *albedo[top_idx] + src[top_idx];

            for (int ilay=nlay-1; ilay >= 0; --ilay)
            {
                    const int lev_idx1 = icol + ilay*ncol + igpt*(nlay+1)*ncol;
                    const int lev_idx2 = icol + (ilay+1)*ncol + igpt*(nlay+1)*ncol;
                    const int lay_idx = icol + ilay*ncol + igpt*nlay*ncol;
                    flux_dn[lev_idx1] = (t_dif[lay_idx]*flux_dn[lev_idx2] +
                                         r_dif[lay_idx]*src[lev_idx1] +
                                         source_dn[lay_idx]) * denom[lay_idx];
                    flux_up[lev_idx1] = flux_dn[lev_idx1] * albedo[lev_idx1] + src[lev_idx1];
            }
            for (int ilay=nlay; ilay >= 0; --ilay)
            {
                const int idx = icol + ilay*ncol + igpt*(nlay+1)*ncol;
                flux_dn[idx] += flux_dir[idx];
            }
        }
    }

    template<typename TF>__device__
    void sw_source_kernel(const int icol, const int igpt,
                          const int ncol, const int nlay, const BOOL_TYPE top_at_1,
                          TF* __restrict__ r_dir, TF* __restrict__ t_dir, TF* __restrict__ t_noscat,
                          const TF* __restrict__ sfc_alb_dir, TF* __restrict__ source_up, TF* __restrict__ source_dn,
                          TF* __restrict__ source_sfc, TF* __restrict__ flux_dir)
    {

        if (top_at_1)
        {
            for (int ilay=0; ilay<nlay; ++ilay)
            {
                const int idx_lay  = icol + ilay*ncol + igpt*nlay*ncol;
                const int idx_lev1 = icol + ilay*ncol + igpt*(nlay+1)*ncol;
                const int idx_lev2 = icol + (ilay+1)*ncol + igpt*(nlay+1)*ncol;
                source_up[idx_lay] = r_dir[idx_lay] * flux_dir[idx_lev1];
                source_dn[idx_lay] = t_dir[idx_lay] * flux_dir[idx_lev1];
                flux_dir[idx_lev2] = t_noscat[idx_lay] * flux_dir[idx_lev1];

            }
            const int sfc_idx = icol + igpt*ncol;
            const int flx_idx = icol + nlay*ncol + igpt*(nlay+1)*ncol;
            source_sfc[sfc_idx] = flux_dir[flx_idx] * sfc_alb_dir[icol];
        }
        else
        {
            for (int ilay=nlay-1; ilay>=0; --ilay)
            {
                const int idx_lay  = icol + ilay*ncol + igpt*nlay*ncol;
                const int idx_lev1 = icol + (ilay)*ncol + igpt*(nlay+1)*ncol;
                const int idx_lev2 = icol + (ilay+1)*ncol + igpt*(nlay+1)*ncol;
                source_up[idx_lay] = r_dir[idx_lay] * flux_dir[idx_lev2];
                source_dn[idx_lay] = t_dir[idx_lay] * flux_dir[idx_lev2];
                flux_dir[idx_lev1] = t_noscat[idx_lay] * flux_dir[idx_lev2];

            }
            const int sfc_idx = icol + igpt*ncol;
            const int flx_idx = icol + igpt*(nlay+1)*ncol;
            source_sfc[sfc_idx] = flux_dir[flx_idx] * sfc_alb_dir[icol];
        }

    }

    template<typename TF>__device__
    void apply_BC_kernel_lw(const int icol, const int igpt, const int isfc, int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const TF* __restrict__ inc_flux, TF* __restrict__ flux_dn)
    {
        const int idx_in  = icol + isfc*ncol + igpt*ncol*(nlay+1);
        const int idx_out = (top_at_1) ? icol + igpt*ncol*(nlay+1) : icol + nlay*ncol + igpt*ncol*(nlay+1);
        flux_dn[idx_out] = inc_flux[idx_in];
    }

    template<typename TF>__global__ //apply_BC_gpt
    void apply_BC_kernel(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const TF* __restrict__ inc_flux, TF* __restrict__ flux_dn)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int igpt = blockIdx.y*blockDim.y + threadIdx.y;
        if ( (icol < ncol) && (igpt < ngpt) )
        {
            if (top_at_1)
            {
                const int idx_out = icol + igpt*ncol*(nlay+1);
                const int idx_in  = icol + igpt*ncol;
                flux_dn[idx_out] = inc_flux[idx_in];
            }
            else
            {
                const int idx_out = icol + nlay*ncol + igpt*ncol*(nlay+1);
                const int idx_in  = icol + igpt*ncol;
                flux_dn[idx_out] = inc_flux[idx_in];
            }
        }
    }

    template<typename TF>__global__
    void apply_BC_kernel(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const TF* __restrict__ inc_flux, const TF* __restrict__ factor, TF* __restrict__ flux_dn)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int igpt = blockIdx.y*blockDim.y + threadIdx.y;
        if ( (icol < ncol) && (igpt < ngpt) )
        {
            if (top_at_1)
            {
                const int idx_out = icol + igpt*ncol*(nlay+1);
                const int idx_in  = icol + igpt*ncol;
                flux_dn[idx_out] = inc_flux[idx_in] * factor[icol];
            }
            else
            {
                const int idx_out = icol + nlay*ncol + igpt*ncol*(nlay+1);
                const int idx_in  = icol + igpt*ncol;
                flux_dn[idx_out] = inc_flux[idx_in] * factor[icol];
            }
        }
    }

    template<typename TF>__global__
    void apply_BC_kernel(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, TF* __restrict__ flux_dn)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int igpt = blockIdx.y*blockDim.y + threadIdx.y;
        if ( (icol < ncol) && (igpt < ngpt) )
        {
            if (top_at_1)
            {
                const int idx_out = icol + igpt*ncol*(nlay+1);
                flux_dn[idx_out] = TF(0.);
            }
            else
            {
                const int idx_out = icol + nlay*ncol + igpt*ncol*(nlay+1);
                flux_dn[idx_out] = TF(0.);
            }
        }
    }

    template<typename TF>__global__
    void sw_2stream_kernel(const int ncol, const int nlay, const int ngpt, const TF tmin,
            const TF* __restrict__ tau, const TF* __restrict__ ssa, const TF* __restrict__ g, const TF* __restrict__ mu0,
            TF* __restrict__ r_dif, TF* __restrict__ t_dif,
            TF* __restrict__ r_dir, TF* __restrict__ t_dir, TF* __restrict__ t_noscat)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
        const int igpt = blockIdx.z*blockDim.z + threadIdx.z;

        if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
        {
            const int idx = icol + ilay*ncol + igpt*nlay*ncol;
            const TF mu0_inv = TF(1.)/mu0[icol];
            const TF gamma1 = (TF(8.) - ssa[idx] * (TF(5.) + TF(3.) * g[idx])) * TF(.25);
            const TF gamma2 = TF(3.) * (ssa[idx] * (TF(1.) -          g[idx])) * TF(.25);
            const TF gamma3 = (TF(2.) - TF(3.) * mu0[icol] *          g[idx]) * TF(.25);
            const TF gamma4 = TF(1.) - gamma3;

            const TF alpha1 = gamma1 * gamma4 + gamma2 * gamma3;
            const TF alpha2 = gamma1 * gamma3 + gamma2 * gamma4;

            const TF k = sqrt(max((gamma1 - gamma2) * (gamma1 + gamma2), TF(1e-12)));
            const TF exp_minusktau = exp(-tau[idx] * k);
            const TF exp_minus2ktau = exp_minusktau * exp_minusktau;

            const TF rt_term = TF(1.) / (k      * (TF(1.) + exp_minus2ktau) +
                                         gamma1 * (TF(1.) - exp_minus2ktau));
            r_dif[idx] = rt_term * gamma2 * (TF(1.) - exp_minus2ktau);
            t_dif[idx] = rt_term * TF(2.) * k * exp_minusktau;
            t_noscat[idx] = exp(-tau[idx] * mu0_inv);

            const TF k_mu     = k * mu0[icol];
            const TF k_gamma3 = k * gamma3;
            const TF k_gamma4 = k * gamma4;

            const TF fact = (abs(TF(1.) - k_mu*k_mu) > tmin) ? TF(1.) - k_mu*k_mu : tmin;
            const TF rt_term2 = ssa[idx] * rt_term / fact;

            r_dir[idx] = rt_term2  * ((TF(1.) - k_mu) * (alpha2 + k_gamma3)   -
                                      (TF(1.) + k_mu) * (alpha2 - k_gamma3) * exp_minus2ktau -
                                      TF(2.) * (k_gamma3 - alpha2 * k_mu)  * exp_minusktau * t_noscat[idx]);

            t_dir[idx] = -rt_term2 * ((TF(1.) + k_mu) * (alpha1 + k_gamma4) * t_noscat[idx]   -
                                      (TF(1.) - k_mu) * (alpha2 - k_gamma4) * exp_minus2ktau * t_noscat[idx] -
                                       TF(2.) * (k_gamma4 + alpha1 * k_mu) * exp_minusktau);
        }
    }

    template<typename TF>__global__
    void sw_source_adding_kernel(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1,
                                 const TF* __restrict__ sfc_alb_dir, const TF* __restrict__ sfc_alb_dif,
                                 TF* __restrict__ r_dif, TF* __restrict__ t_dif,
                                 TF* __restrict__ r_dir, TF* __restrict__ t_dir, TF* __restrict__ t_noscat,
                                 TF* __restrict__ flux_up, TF* __restrict__ flux_dn, TF* __restrict__ flux_dir,
                                 TF* __restrict__ source_up, TF* __restrict__ source_dn, TF* __restrict__ source_sfc,
                                 TF* __restrict__ albedo, TF* __restrict__ src, TF* __restrict__ denom)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int igpt = blockIdx.y*blockDim.y + threadIdx.y;

        if ( (icol < ncol) && (igpt < ngpt) )
        {
            sw_source_kernel(icol, igpt, ncol, nlay, top_at_1, r_dir, t_dir,
                             t_noscat, sfc_alb_dir, source_up, source_dn, source_sfc, flux_dir);

            sw_adding_kernel(icol, igpt, ncol, nlay, top_at_1, sfc_alb_dif,
                             r_dif, t_dif, source_dn, source_up, source_sfc,
                             flux_up, flux_dn, flux_dir, albedo, src, denom);
        }
    }

    template<typename TF>__global__
    void lw_solver_noscat_gaussquad_kernel(const int ncol, const int nlay, const int ngpt, const TF eps,
                                           const BOOL_TYPE top_at_1, const int nmus, const TF* __restrict__ ds, const TF* __restrict__ weights,
                                           const TF* __restrict__ tau, const TF* __restrict__ lay_source,
                                           const TF* __restrict__ lev_source_inc, const TF* __restrict__ lev_source_dec, const TF* __restrict__ sfc_emis,
                                           const TF* __restrict__ sfc_src, TF* __restrict__ radn_up, TF* __restrict__ radn_dn,
                                           const TF* __restrict__ sfc_src_jac, TF* __restrict__ radn_up_jac, TF* __restrict__ tau_loc,
                                           TF* __restrict__ trans, TF* __restrict__ source_dn, TF* __restrict__ source_up,
                                           TF* __restrict__ source_sfc, TF* __restrict__ sfc_albedo, TF* __restrict__ source_sfc_jac,
                                           TF* __restrict__ flux_up, TF* __restrict__ flux_dn, TF* __restrict__ flux_up_jac)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int igpt = blockIdx.y*blockDim.y + threadIdx.y;

        if ( (icol < ncol) && (igpt < ngpt) )
        {
            lw_solver_noscat_kernel(icol, igpt, ncol, nlay, ngpt, eps, top_at_1, ds[0], weights[0], tau, lay_source,
                             lev_source_inc, lev_source_dec, sfc_emis, sfc_src, flux_up, flux_dn, sfc_src_jac,
                             flux_up_jac, tau_loc, trans, source_dn, source_up, source_sfc, sfc_albedo, source_sfc_jac);
            const int top_level = top_at_1 ? 0 : nlay;
            apply_BC_kernel_lw(icol, igpt, top_level, ncol, nlay, ngpt, top_at_1, flux_dn, radn_dn);

            if (nmus > 1)
            {
                for (int imu=1; imu<nmus; ++imu)
                {
                    lw_solver_noscat_kernel(icol, igpt, ncol, nlay, ngpt, eps, top_at_1, ds[imu], weights[imu], tau, lay_source,
                                     lev_source_inc, lev_source_dec, sfc_emis, sfc_src, radn_up, radn_dn, sfc_src_jac,
                                     radn_up_jac, tau_loc, trans, source_dn, source_up, source_sfc, sfc_albedo, source_sfc_jac);

                    for (int ilev=0; ilev<(nlay+1); ++ilev)
                    {
                        const int idx = icol + ilev*ncol + igpt*ncol*(nlay+1);
                        flux_up[idx] += radn_up[idx];
                        flux_dn[idx] += radn_dn[idx];
                        flux_up_jac[idx] += radn_up_jac[idx];
                    }
                }
            }
        }
    }
}

namespace rte_kernel_launcher_cuda
{
    template<typename TF>
    void apply_BC(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1,
                  const Array_gpu<TF,2>& inc_flux_dir, const Array_gpu<TF,1>& mu0, Array_gpu<TF,3>& gpt_flux_dir)
    {
        const int block_col = 32;
        const int block_gpt = 32;

        const int grid_col  = ncol/block_col + (ncol%block_col > 0);
        const int grid_gpt  = ngpt/block_gpt + (ngpt%block_gpt > 0);

        dim3 grid_gpu(grid_col, grid_gpt);
        dim3 block_gpu(block_col, block_gpt);
        apply_BC_kernel<<<grid_gpu, block_gpu>>>(ncol, nlay, ngpt, top_at_1, inc_flux_dir.ptr(), mu0.ptr(), gpt_flux_dir.ptr());

    }

    template<typename TF>
    void apply_BC(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, Array_gpu<TF,3>& gpt_flux_dn)
    {
        const int block_col = 32;
        const int block_gpt = 32;

        const int grid_col  = ncol/block_col + (ncol%block_col > 0);
        const int grid_gpt  = ngpt/block_gpt + (ngpt%block_gpt > 0);

        dim3 grid_gpu(grid_col, grid_gpt);
        dim3 block_gpu(block_col, block_gpt);
        apply_BC_kernel<<<grid_gpu, block_gpu>>>(ncol, nlay, ngpt, top_at_1, gpt_flux_dn.ptr());
    }

    template<typename TF>
    void apply_BC(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const Array_gpu<TF,2>& inc_flux_dif, Array_gpu<TF,3>& gpt_flux_dn)
    {
        const int block_col = 32;
        const int block_gpt = 32;

        const int grid_col  = ncol/block_col + (ncol%block_col > 0);
        const int grid_gpt  = ngpt/block_gpt + (ngpt%block_gpt > 0);

        dim3 grid_gpu(grid_col, grid_gpt);
        dim3 block_gpu(block_col, block_gpt);
        apply_BC_kernel<<<grid_gpu, block_gpu>>>(ncol, nlay, ngpt, top_at_1, inc_flux_dif.ptr(), gpt_flux_dn.ptr());
    }

    template<typename TF>
    void lw_solver_noscat_gaussquad(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const int nmus,
                                    const Array_gpu<TF,2>& ds, const Array_gpu<TF,2>& weights, const Array_gpu<TF,3>& tau, const Array_gpu<TF,3> lay_source,
                                    const Array_gpu<TF,3>& lev_source_inc, const Array_gpu<TF,3>& lev_source_dec, const Array_gpu<TF,2>& sfc_emis,
                                    const Array_gpu<TF,2>& sfc_src, Array_gpu<TF,3>& flux_up, Array_gpu<TF,3>& flux_dn,
                                    const Array_gpu<TF,2>& sfc_src_jac, Array_gpu<TF,3>& flux_up_jac)
    {
        float elapsedtime;
        TF eps = std::numeric_limits<TF>::epsilon();
        const int flx_size = flux_dn.size() * sizeof(TF);
        const int opt_size = tau.size() * sizeof(TF);
        const int mus_size = nmus * sizeof(TF);
        const int sfc_size = sfc_src.size() * sizeof(TF);

        TF* tau_loc;
        TF* radn_up;
        TF* radn_up_jac;
        TF* radn_dn;
        TF* trans;
        TF* source_dn;
        TF* source_up;
        TF* source_sfc;
        TF* source_sfc_jac;
        TF* sfc_albedo;

        cuda_safe_call(hipMalloc((void **) &source_sfc, sfc_size));
        cuda_safe_call(hipMalloc((void **) &source_sfc_jac, sfc_size));
        cuda_safe_call(hipMalloc((void **) &sfc_albedo, sfc_size));
        cuda_safe_call(hipMalloc((void **) &tau_loc, opt_size));
        cuda_safe_call(hipMalloc((void **) &trans, opt_size));
        cuda_safe_call(hipMalloc((void **) &source_dn, opt_size));
        cuda_safe_call(hipMalloc((void **) &source_up, opt_size));
        cuda_safe_call(hipMalloc((void **) &radn_dn, flx_size));
        cuda_safe_call(hipMalloc((void **) &radn_up, flx_size));
        cuda_safe_call(hipMalloc((void **) &radn_up_jac, flx_size));

        const int block_col2d = 32;
        const int block_gpt2d = 1;

        const int grid_col2d  = ncol/block_col2d + (ncol%block_col2d > 0);
        const int grid_gpt2d  = ngpt/block_gpt2d + (ngpt%block_gpt2d > 0);

        dim3 grid_gpu2d(grid_col2d, grid_gpt2d);
        dim3 block_gpu2d(block_col2d, block_gpt2d);
        lw_solver_noscat_gaussquad_kernel<<<grid_gpu2d, block_gpu2d>>>(
                ncol, nlay, ngpt, eps, top_at_1, nmus, ds.ptr(), weights.ptr(), tau.ptr(), lay_source.ptr(),
                lev_source_inc.ptr(), lev_source_dec.ptr(), sfc_emis.ptr(), sfc_src.ptr(), radn_up,
                radn_dn, sfc_src_jac.ptr(), radn_up_jac, tau_loc, trans, source_dn, source_up,
                source_sfc, sfc_albedo, source_sfc_jac, flux_up.ptr(), flux_dn.ptr(), flux_up_jac.ptr());

        cuda_safe_call(hipFree(tau_loc));
        cuda_safe_call(hipFree(radn_up));
        cuda_safe_call(hipFree(radn_up_jac));
        cuda_safe_call(hipFree(radn_dn));
        cuda_safe_call(hipFree(trans));
        cuda_safe_call(hipFree(source_dn));
        cuda_safe_call(hipFree(source_up));
        cuda_safe_call(hipFree(source_sfc));
        cuda_safe_call(hipFree(source_sfc_jac));
        cuda_safe_call(hipFree(sfc_albedo));
    }

    template<typename TF>
    void sw_solver_2stream(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1,
                           const Array_gpu<TF,3>& tau, const Array_gpu<TF,3>& ssa, const Array_gpu<TF,3>& g,
                           const Array_gpu<TF,1>& mu0, const Array_gpu<TF,2>& sfc_alb_dir, const Array_gpu<TF,2>& sfc_alb_dif,
                           Array_gpu<TF,3>& flux_up, Array_gpu<TF,3>& flux_dn, Array_gpu<TF,3>& flux_dir)
    {
        const int opt_size = tau.size() * sizeof(TF);
        const int alb_size  = sfc_alb_dir.size() * sizeof(TF);
        const int flx_size  = flux_up.size() * sizeof(TF);
        TF* r_dif;
        TF* t_dif;
        TF* r_dir;
        TF* t_dir;
        TF* t_noscat;
        TF* source_up;
        TF* source_dn;
        TF* source_sfc;
        TF* albedo;
        TF* src;
        TF* denom;

        cuda_safe_call(hipMalloc((void **) &r_dif, opt_size));
        cuda_safe_call(hipMalloc((void **) &t_dif, opt_size));
        cuda_safe_call(hipMalloc((void **) &r_dir, opt_size));
        cuda_safe_call(hipMalloc((void **) &t_dir, opt_size));
        cuda_safe_call(hipMalloc((void **) &t_noscat, opt_size));
        cuda_safe_call(hipMalloc((void **) &source_up, opt_size));
        cuda_safe_call(hipMalloc((void **) &source_dn, opt_size));
        cuda_safe_call(hipMalloc((void **) &source_sfc, alb_size));
        cuda_safe_call(hipMalloc((void **) &albedo, flx_size));
        cuda_safe_call(hipMalloc((void **) &src, flx_size));
        cuda_safe_call(hipMalloc((void **) &denom, opt_size));
        const int block_col3d = 32;
        const int block_lay3d = 16;
        const int block_gpt3d = 1;

        const int grid_col3d  = ncol/block_col3d + (ncol%block_col3d > 0);
        const int grid_lay3d  = nlay/block_lay3d + (nlay%block_lay3d > 0);
        const int grid_gpt3d  = ngpt/block_gpt3d + (ngpt%block_gpt3d > 0);

        dim3 grid_gpu3d(grid_col3d, grid_lay3d, grid_gpt3d);
        dim3 block_gpu3d(block_col3d, block_lay3d, block_gpt3d);

        TF tmin = std::numeric_limits<TF>::epsilon();
        sw_2stream_kernel<<<grid_gpu3d, block_gpu3d>>>(
                ncol, nlay, ngpt, tmin, tau.ptr(), ssa.ptr(), g.ptr(), mu0.ptr(), r_dif, t_dif, r_dir, t_dir, t_noscat);

        const int block_col2d = 32;
        const int block_gpt2d = 32;

        const int grid_col2d  = ncol/block_col2d + (ncol%block_col2d > 0);
        const int grid_gpt2d  = ngpt/block_gpt2d + (ngpt%block_gpt2d > 0);

        dim3 grid_gpu2d(grid_col2d, grid_gpt2d);
        dim3 block_gpu2d(block_col2d, block_gpt2d);
        sw_source_adding_kernel<<<grid_gpu2d, block_gpu2d>>>(
                ncol, nlay, ngpt, top_at_1, sfc_alb_dir.ptr(), sfc_alb_dif.ptr(), r_dif, t_dif, r_dir, t_dir, t_noscat,
                flux_up.ptr(), flux_dn.ptr(), flux_dir.ptr(), source_up, source_dn, source_sfc, albedo, src, denom);

        cuda_safe_call(hipFree(r_dif));
        cuda_safe_call(hipFree(t_dif));
        cuda_safe_call(hipFree(r_dir));
        cuda_safe_call(hipFree(t_dir));
        cuda_safe_call(hipFree(t_noscat));
        cuda_safe_call(hipFree(source_up));
        cuda_safe_call(hipFree(source_dn));
        cuda_safe_call(hipFree(source_sfc));
        cuda_safe_call(hipFree(albedo));
        cuda_safe_call(hipFree(src));
        cuda_safe_call(hipFree(denom));
    }
}

#ifdef FLOAT_SINGLE_RRTMGP
template void rte_kernel_launcher_cuda::sw_solver_2stream<float>(
            const int, const int, const int, const BOOL_TYPE,
            const Array_gpu<float,3>&, const Array_gpu<float,3>&, const Array_gpu<float,3>&,
            const Array_gpu<float,1>&, const Array_gpu<float,2>&, const Array_gpu<float,2>&,
            Array_gpu<float,3>&, Array_gpu<float,3>&, Array_gpu<float,3>&);

template void rte_kernel_launcher_cuda::lw_solver_noscat_gaussquad<float>(
            const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const int nmus,
            const Array_gpu<float,2>& ds, const Array_gpu<float,2>& weights, const Array_gpu<float,3>& tau, const Array_gpu<float,3> lay_source,
            const Array_gpu<float,3>& lev_source_inc, const Array_gpu<float,3>& lev_source_dec, const Array_gpu<float,2>& sfc_emis,
            const Array_gpu<float,2>& sfc_src, Array_gpu<float,3>& flux_dn, Array_gpu<float,3>& flux_up,
            const Array_gpu<float,2>& sfc_src_jac, Array_gpu<float,3>& flux_up_jac);

#else
template void rte_kernel_launcher_cuda::apply_BC(const int, const int, const int, const BOOL_TYPE,
                  const Array_gpu<double,2>&, const Array_gpu<double,1>&, Array_gpu<double,3>&);
template void rte_kernel_launcher_cuda::apply_BC(const int, const int, const int, const BOOL_TYPE, Array_gpu<double,3>&);
template void rte_kernel_launcher_cuda::apply_BC(const int, const int, const int, const BOOL_TYPE,
                  const Array_gpu<double,2>&, Array_gpu<double,3>&);


template void rte_kernel_launcher_cuda::sw_solver_2stream<double>(
            const int, const int, const int, const BOOL_TYPE,
            const Array_gpu<double,3>&, const Array_gpu<double,3>&, const Array_gpu<double,3>&,
            const Array_gpu<double,1>&, const Array_gpu<double,2>&, const Array_gpu<double,2>&,
            Array_gpu<double,3>&, Array_gpu<double,3>&, Array_gpu<double,3>&);

template void rte_kernel_launcher_cuda::lw_solver_noscat_gaussquad<double>(
            const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const int nmus,
            const Array_gpu<double,2>& ds, const Array_gpu<double,2>& weights, const Array_gpu<double,3>& tau, const Array_gpu<double,3> lay_source,
            const Array_gpu<double,3>& lev_source_inc, const Array_gpu<double,3>& lev_source_dec, const Array_gpu<double,2>& sfc_emis,
            const Array_gpu<double,2>& sfc_src, Array_gpu<double,3>& flux_up, Array_gpu<double,3>& flux_dn,
            const Array_gpu<double,2>& sfc_src_jac,Array_gpu<double,3>& flux_up_jac);
#endif
